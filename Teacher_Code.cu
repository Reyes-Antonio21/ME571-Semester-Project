#include "hip/hip_runtime.h"
# include "common.h"
# include <hip/hip_runtime.h>
# include <stdlib.h>
# include <stdio.h>
# include <math.h>
# include <string.h>
# include <time.h>

#define ID_2D(i,j,nx) ((i)*(nx+2)+(j))

int main ( int argc, char *argv[] );
void initial_conditions ( int nx, int ny, float dx, float dy,  float x_length, float x[],float y[], float h[], float uh[] ,float vh[]);

//utilities
void getArgs(int *nx, float *dt, float *x_length, float *t_final, int argc, char *argv[]);

void write_results ( char *output_filename, int nx, int ny, float x[], float y[], float h[], float uh[], float vh[]);


__global__ void computeFluxesGPU(float *h,  float *uh,  float *vh, 
				 float *fh, float *fuh, float *fvh,
				 float *gh, float *guh, float *gvh,
				 int nx, int ny)
{
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int id;

    float g=9.81;

      // **** COMPUTE FLUXES ****
      //Compute fluxes (including ghosts) 

    if (i < nx+2 && j < ny+2){
	  id=ID_2D(i,j,nx);

	  fh[id] = uh[id]; //flux for the height equation: u*h
	  fuh[id] = uh[id]*uh[id]/h[id] + 0.5*g*h[id]*h[id]; //flux for the momentum equation: u^2*h + 0.5*g*h^2
	  fvh[id] = uh[id]*vh[id]/h[id]; //flux for the momentum equation: u*v**h 
	  gh[id] = vh[id]; //flux for the height equation: v*h
	  guh[id] = uh[id]*vh[id]/h[id]; //flux for the momentum equation: u*v**h 
	  gvh[id] = vh[id]*vh[id]/h[id] + 0.5*g*h[id]*h[id]; //flux for the momentum equation: v^2*h + 0.5*g*h^2
	}

}


/******************************************************************************/

int main ( int argc, char *argv[] )

/******************************************************************************/
/*
  Purpose:
    MAIN is the main program for SHALLOW_WATER_2D.

  Discussion:
    SHALLOW_WATER_2D approximates the 2D shallow water equations.
    The version of the shallow water equations being solved here is in
    conservative form, and omits the Coriolis force.  The state variables
    are H (the height) and UH (the mass velocity).

    The equations have the form
      dH/dt + d UH/dx = 0
      d UH/dt + d ( U^2 H + 1/2 g H^2 )/dx + d ( U V H             )/dy = 0
      d VH/dt + d ( U V H             )/dx + d ( V^2 H + 1/2 g H^2 )/dy = 0

    Here U is the ordinary velocity, U = UH/H, and g is the gravitational
    acceleration.
    The initial conditions are used to specify ( H, UH ) at an equally
    spaced set of points, and then the Lax-Friedrichs method is used to advance
    the solution until a final time t_final, with
    boundary conditions supplying the first and last spatial values.
    Some input values will result in an unstable calculation that
    quickly blows up.  This is related to the Courant-Friedrichs-Levy
    condition, which requires that DT be small enough, relative to DX and
    the velocity, that information cannot cross an entire cell.

    A "reasonable" set of input quantities is
      sw_2d 401 0.002 10.0 0.2

  Licensing:
    This code is distributed under the GNU LGPL license.

  Modified:
    26 March 2019 by Michal A. Kopera
    20 April 2022 by Michal A. Kopera

  Author:
    John Burkardt

  Reference:
    Cleve Moler,
    "The Shallow Water Equations",
    Experiments with MATLAB.

  Parameters:
    Input, integer NX, the number of spatial nodes.
    Input, integer DT, the size of a time step.
    Input, real X_LENGTH, the length of the region.
    Input, real T_FINAL, the final time of simulation.

    Output, real X[NX], the X coordinates.
    Output, real H[NX], the height for all space points at time t_final.
    Output, real UH[NX], the mass velocity (discharge) for all space points at time t_final.
*/

{
  float dx;
  float dy;
  float dt;
  float g = 9.81; //[m^2/s] gravitational constant
  float *h;
  float *fh, *h_fh;
  float *gh, *h_gh;
  float *hm;
  int i,j, id, id_left, id_right, id_bottom, id_top;
  int nx, ny;
  float t_final;
  float *uh;
  float *fuh, *h_fuh;
  float *guh, *h_guh;
  float *uhm;
  float *vh;
  float *fvh, *h_fvh;
  float *gvh, *h_gvh;
  float *vhm;
  float *x;
  float *y;
  float x_length, time;

  //printf ( "\n" );
  //printf ( "SHALLOW_WATER_2D\n" );
  //printf ( "\n" );

  //get command line arguments
  getArgs(&nx, &dt, &x_length, &t_final, argc, argv);
  
 
  //printf ( "  NX = %d\n", nx );
  //printf ( "  DT = %g\n", dt );
  //printf ( "  X_LENGTH = %g\n", x_length );
  //printf ( "  T_FINAL = %g\n", t_final );
  
  ny=nx; // we assume this, does not have to be this way

  // **** ALLOCATE MEMORY ****
  
  //Allocate space (nx+2)((nx+2) long, to accound for ghosts
  //height array
  h  = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  hm = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  fh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  gh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  h_fh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  h_gh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );

  //x momentum array
  uh  = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  uhm = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  fuh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  guh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );

  h_fuh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  h_guh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  //y momentum array
  vh  = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  vhm = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  fvh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  gvh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );

  h_fvh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  h_gvh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  // location arrays
  x = ( float * ) malloc ( nx * sizeof ( float ) );
  y = ( float * ) malloc ( ny * sizeof ( float ) );

  //Allocate memory on the device
  float *d_h, *d_uh, *d_vh;
  float *d_fh, *d_fuh, *d_fvh;
  float *d_gh, *d_guh, *d_gvh;

  CHECK(hipMalloc((void **)&d_h, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_uh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_vh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_fh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_fuh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_fvh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_gh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_guh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_gvh, (nx+2)*(ny+2) * sizeof ( float )));

  //Define the locations of the nodes and time steps and the spacing.
  dx = x_length / ( float ) ( nx );
  dy = x_length / ( float ) ( nx );

    // **** INITIAL CONDITIONS ****
  //Apply the initial conditions.
  //printf("Before initial conditions\n");
  initial_conditions ( nx, ny, dx, dy, x_length,  x, y, h, uh, vh);

  //printf("Before write results\n");
  //Write initial condition to a file
  write_results("swe2d_cuda_init.dat",nx,ny,x,y,h,uh,vh);


  // **** TIME LOOP ****
  float lambda_x = 0.5*dt/dx;
  float lambda_y = 0.5*dt/dy;


  time=0;
  int k=0; //time-step counter
  //start timer
  clock_t time_start = clock();

  while (time<t_final) //time loop begins
    {
      //  Take a time step
      time=time+dt;
      k++;
      //printf("time = %f\n",time);
      // **** COMPUTE FLUXES ****
      //Compute fluxes (including ghosts) 
      /*      for ( i = 0; i < ny+2; i++ )
	for ( j = 0; j < nx+2; j++){
	  id=ID_2D(i,j,nx);

	  fh[id] = uh[id]; //flux for the height equation: u*h
	  fuh[id] = uh[id]*uh[id]/h[id] + 0.5*g*h[id]*h[id]; //flux for the momentum equation: u^2*h + 0.5*g*h^2
	  fvh[id] = uh[id]*vh[id]/h[id]; //flux for the momentum equation: u*v**h 
	  gh[id] = vh[id]; //flux for the height equation: v*h
	  guh[id] = uh[id]*vh[id]/h[id]; //flux for the momentum equation: u*v**h 
	  gvh[id] = vh[id]*vh[id]/h[id] + 0.5*g*h[id]*h[id]; //flux for the momentum equation: v^2*h + 0.5*g*h^2
	}
      */
      //Move data to the device
      CHECK(hipMemcpy(d_h, h, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyHostToDevice));
      CHECK(hipMemcpy(d_uh, uh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyHostToDevice));
      CHECK(hipMemcpy(d_vh, vh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyHostToDevice));

      
      int dimx = 32;
      int dimy = 32;
      dim3 block(dimx, dimy);
      dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

      computeFluxesGPU<<<grid, block>>>(d_h, d_uh, d_vh,
					d_fh, d_fuh, d_fvh,
					d_gh, d_guh, d_gvh,
					nx, ny);
      CHECK(hipGetLastError());

      //Move fluxes back - for now
      CHECK(hipMemcpy(fh, d_fh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyDeviceToHost));
      CHECK(hipMemcpy(fuh, d_fuh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyDeviceToHost));
      CHECK(hipMemcpy(fvh, d_fvh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyDeviceToHost));
      CHECK(hipMemcpy(gh, d_gh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyDeviceToHost));
      CHECK(hipMemcpy(guh, d_guh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyDeviceToHost));
      CHECK(hipMemcpy(gvh, d_gvh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyDeviceToHost));
      


      // **** COMPUTE VARIABLES ****
      //Compute updated variables
      for ( i = 1; i < ny+1; i++ )
	      for ( j = 1; j < nx+1; j++ )
        {
          id=ID_2D(i,j,nx);
          id_left=ID_2D(i,j-1,nx);
          id_right=ID_2D(i,j+1,nx);
          id_bottom=ID_2D(i-1,j,nx);
          id_top=ID_2D(i+1,j,nx);
          hm[id] = 0.25*(h[id_left]+h[id_right]+h[id_bottom]+h[id_top]) 
            - lambda_x * ( fh[id_right] - fh[id_left] ) 
            - lambda_y * ( gh[id_top] - gh[id_bottom] );
          uhm[id] = 0.25*(uh[id_left]+uh[id_right]+uh[id_bottom]+uh[id_top]) 
            - lambda_x * ( fuh[id_right] - fuh[id_left] ) 
            - lambda_y * ( guh[id_top] - guh[id_bottom] );
          vhm[id] = 0.25*(vh[id_left]+vh[id_right]+vh[id_bottom]+vh[id_top]) 
            - lambda_x * ( fvh[id_right] - fvh[id_left] ) 
            - lambda_y * ( gvh[id_top] - gvh[id_bottom] );
        }

      // **** UPDATE VARIABLES ****
      //update interior state variables
      for (i = 1; i < ny+1; i++)
	for (j = 1; j < nx+1; j++){
	  id=ID_2D(i,j,nx);
	  h[id] = hm[id];
	  uh[id] = uhm[id];
	  vh[id] = vhm[id];
      }

      // **** APPLY BOUNDARY CONDITIONS ****
      //Update the ghosts (boundary conditions)
      //left
      j=1;
      for(i=1; i<ny+1; i++){
	id = ID_2D(i,j,nx);
	id_left = ID_2D(i,j-1,nx);
	h[id_left]  =   h[id];
	uh[id_left] = - uh[id];
	vh[id_left] =   vh[id];
      }

      //right
      j=nx;
      for(i=1; i<ny+1; i++){
	id = ID_2D(i,j,nx);
	id_right = ID_2D(i,j+1,nx);
	h[id_right]  =   h[id];
	uh[id_right] = - uh[id];
	vh[id_right] =   vh[id];
      }

      //bottom
      i=1;
      for(j=1; j<nx+1; j++){
	id = ID_2D(i,j,nx);
	id_bottom = ID_2D(i-1,j,nx);
	h[id_bottom]  =   h[id];
	uh[id_bottom] =   uh[id];
	vh[id_bottom] = - vh[id];
      }

      //top
      i=ny;
      for(j=1; j<nx+1; j++){
	id = ID_2D(i,j,nx);
	id_top = ID_2D(i+1,j,nx);
	h[id_top]  =   h[id];
	uh[id_top] =   uh[id];
	vh[id_top] = - vh[id];
      }

    } //end time loop

clock_t time_end = clock();
double time_elapsed = (double)(time_end - time_start) / CLOCKS_PER_SEC;
  
 printf("Problem size: %d, time steps taken: %d,  elapsed time: %f s\n", nx,k,time_elapsed);
  
  // **** POSTPROCESSING ****
  // Write data to file
  write_results("sw2d_cuda_final.dat",nx,ny,x,y,h,uh,vh);

  CHECK(hipFree(d_h));
  CHECK(hipFree(d_uh));
  CHECK(hipFree(d_vh));
  CHECK(hipFree(d_fh));
  CHECK(hipFree(d_fuh));
  CHECK(hipFree(d_fvh));
  CHECK(hipFree(d_gh));
  CHECK(hipFree(d_guh));
  CHECK(hipFree(d_gvh));


  //Free memory.
  free ( h );
  free ( uh );
  free ( vh ); 
  free ( fh );
  free ( fuh );
  free ( fvh ); 
  free ( gh );
  free ( guh );
  free ( gvh ); 

  free ( h_fh );
  free ( h_fuh );
  free ( h_fvh ); 
  free ( h_gh );
  free ( h_guh );
  free ( h_gvh ); 

  free ( x );
  free ( y );

 //Terminate.

  //printf ( "\n" );
  //printf ( "SHALLOW_WATER_2D:\n" );
  //printf ( "  Normal end of execution.\n" );
  //printf ( "\n" );

  return 0;
}
/******************************************************************************/

void initial_conditions ( int nx, int ny, float dx, float dy,  float x_length, float x[],float y[], float h[], float uh[] ,float vh[]){
  int i,j, id, id1;

  for ( i = 1; i < nx+1; i++ )
    {
      x[i-1] = -x_length/2+dx/2+(i-1)*dx;
      y[i-1] = -x_length/2+dy/2+(i-1)*dy;
    }

  for ( i = 1; i < nx+1; i++ )
    for( j = 1; j < ny+1; j++)
      {
	float xx = x[j-1];
	float yy = y[i-1];
	id=ID_2D(i,j,nx);
	h[id] = 1.0 + 0.4*exp ( -5 * ( xx*xx + yy*yy) );
      }
  
  for ( i = 1; i < nx+1; i++ )
    for( j = 1; j < ny+1; j++)
      {
	id=ID_2D(i,j,nx);
	uh[id] = 0.0;
	vh[id] = 0.0;
      }

  //set boundaries
  //bottom
  i=0;
  for( j = 1; j < nx+1; j++)
    {
      id=ID_2D(i,j,nx);
      id1=ID_2D(i+1,j,nx);

      h[id] = h[id1];
      uh[id] = 0.0;
      vh[id] = 0.0;
    }

  //top
  i=nx+1;
  for( j = 1; j < nx+1; j++)
    {
      id=ID_2D(i,j,nx);
      id1=ID_2D(i-1,j,nx);

      h[id] = h[id1];
      uh[id] = 0.0;
      vh[id] = 0.0;
    }

  //left
  j=0;
  for( i = 1; i < ny+1; i++)
    {
      id=ID_2D(i,j,nx);
      id1=ID_2D(i,j+1,nx);

      h[id] = h[id1];
      uh[id] = 0.0;
      vh[id] = 0.0;
    }

  //right
  j=nx+1;
  for( i = 1; i < ny+1; i++)
    {
      id=ID_2D(i,j,nx);
      id1=ID_2D(i,j-1,nx);

      h[id] = h[id1];
      uh[id] = 0.0;
      vh[id] = 0.0;
    }

  return;
}
/******************************************************************************/


void write_results ( char *output_filename, int nx, int ny, float x[], float y[], float h[], float uh[], float vh[])
/******************************************************************************/

{
  int i,j, id;
  FILE *output;
   
  //Open the file.
  output = fopen ( output_filename, "wt" );
    
  if ( !output ){
    fprintf ( stderr, "\n" );
    fprintf ( stderr, "WRITE_RESULTS - Fatal error!\n" );
    fprintf ( stderr, "  Could not open the output file.\n" );
    exit ( 1 );
  }
    
  //Write the data.
  for ( i = 0; i < ny; i++ ) 
    for ( j = 0; j < nx; j++ ){
        id=ID_2D(i+1,j+1,nx);
	fprintf ( output, "  %24.16g\t%24.16g\t%24.16g\t %24.16g\t %24.16g\n", x[j], y[i],h[id], uh[id], vh[id]);
      }
    
  //Close the file.
  fclose ( output );
  
  return;
}
/******************************************************************************/

void getArgs(int *nx, float *dt, float *x_length, float *t_final, int argc, char *argv[])
{

  /*
    Get the quadrature file root name:
  */
  if ( argc <= 1 ){
    *nx = 401;
  }else{
    *nx = atoi ( argv[1] );
  }
  
  if ( argc <= 2 ){
    *dt = 0.002;
  }else{
    *dt = atof ( argv[2] );
  }
  
  if ( argc <= 3 ){
    *x_length = 10.0;
  }else{
    *x_length = atof ( argv[3] );
    }
  
  if ( argc <= 4 ){
    *t_final = 0.5;
  }else{
    *t_final = atof ( argv[4] );
  }
}