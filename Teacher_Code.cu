#include "hip/hip_runtime.h"
# include "common.h"
# include <stdlib.h>
# include <stdio.h>
# include <math.h>
# include <string.h>
# include <time.h>
# include <hip/hip_runtime.h>

#define ID_2D(i,j,nx) ((i)*(nx+2)+(j))

#define EPSILON 1e-6f  // Small value to prevent division by zero

//************************************************ UTILITIES ************************************************//

void getArgs(int *nx, double *dt, float *x_length, float *t_final, int argc, char *argv[])
{
  // Get the quadrature file root name:

  if ( argc <= 1 ){
    *nx = 401;
  }else{
    *nx = atoi ( argv[1] );
  }
  
  if ( argc <= 2 ){
    *dt = 0.002;
  }else{
    *dt = atof ( argv[2] );
  }
  
  if ( argc <= 3 ){
    *x_length = 10.0;
  }else{
    *x_length = atof ( argv[3] );
    }
  
  if ( argc <= 4 ){
    *t_final = 0.5;
  }else{
    *t_final = atof ( argv[4] );
  }
}
/******************************************************************************/

void writeResults(float h[], float uh[], float vh[], float x[], float y[], float time, int nx, int ny)
{
  char filename[50];

  int i, j, id;

  //Create the filename based on the time step.
  sprintf(filename, "tc2d_%08.6f.dat", time);

  //Open the file.
  FILE *file = fopen (filename, "wt" );
    
  if (!file)
  {
    fprintf (stderr, "\n" );

    fprintf (stderr, "WRITE_RESULTS - Fatal error!\n");

    fprintf (stderr, "  Could not open the output file.\n");

    exit (1);
  }

  else
  {  
    //Write the data.
    for ( i = 0; i < ny; i++ ) 
      for ( j = 0; j < nx; j++ )
      {
        id = ID_2D(i + 1, j + 1, nx);
        fprintf ( file, "%24.16g\t%24.16g\t%24.16g\t %24.16g\t %24.16g\n", x[j], y[i], h[id], uh[id], vh[id]);
      }
    
    //Close the file.
    fclose (file);
  }

  return;
}
/******************************************************************************/

void initial_conditions ( int nx, int ny, float dx, float dy,  float x_length, float x[],float y[], float h[], float uh[] ,float vh[])
{
  int i,j, id, id1;

  for ( i = 1; i < nx+1; i++ )
    {
      x[i-1] = -x_length/2+dx/2+(i-1)*dx;
      y[i-1] = -x_length/2+dy/2+(i-1)*dy;
    }

  for ( i = 1; i < nx+1; i++ )
    for( j = 1; j < ny+1; j++)
    {
      float xx = x[j-1];
      float yy = y[i-1];
      id=ID_2D(i,j,nx);
      h[id] = 1.0 + 0.4*exp ( -5 * ( xx*xx + yy*yy) );
    }
  
  for ( i = 1; i < nx+1; i++ )
    for( j = 1; j < ny+1; j++)
    {
      id=ID_2D(i,j,nx);
      uh[id] = 0.0;
      vh[id] = 0.0;
    }

  //set boundaries
  //bottom
  i=0;
  for( j = 1; j < nx+1; j++)
    {
      id=ID_2D(i,j,nx);
      id1=ID_2D(i+1,j,nx);

      h[id] = h[id1];
      uh[id] = 0.0;
      vh[id] = 0.0;
    }

  //top
  i=nx+1;
  for( j = 1; j < nx+1; j++)
    {
      id=ID_2D(i,j,nx);
      id1=ID_2D(i-1,j,nx);

      h[id] = h[id1];
      uh[id] = 0.0;
      vh[id] = 0.0;
    }

  //left
  j=0;
  for( i = 1; i < ny+1; i++)
    {
      id=ID_2D(i,j,nx);
      id1=ID_2D(i,j+1,nx);

      h[id] = h[id1];
      uh[id] = 0.0;
      vh[id] = 0.0;
    }

  //right
  j=nx+1;
  for( i = 1; i < ny+1; i++)
    {
      id=ID_2D(i,j,nx);
      id1=ID_2D(i,j-1,nx);

      h[id] = h[id1];
      uh[id] = 0.0;
      vh[id] = 0.0;
    }

  return;
}
/******************************************************************************/

__global__ void initialConditionsGPU( int nx, int ny, float dx, float dy,  float x_length, float x[],float y[], float h[], float uh[] ,float vh[])
{
  unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned int id, id_boundary;

  if (i > 0 && i < ny + 1)
  {
    x[i - 1] = -x_length / 2 + dx / 2 + (i - 1) * dx;
    y[i - 1] = -x_length / 2 + dy / 2 + (i - 1) * dy; 
  }

  if ( i > 0 && i < ny + 1 && j > 0 && j < nx + 1)
  {
    id = ID_2D(i, j, nx);

    float xx = x[j - 1];
    float yy = y[i - 1];

    h[id] = 1.0 + 0.4 * exp( -5 * ( xx * xx + yy * yy) );
  }
  
  if (i > 0 && i < ny + 1 && j > 0 && j < nx + 1)
  {
    id = ID_2D(i, j, nx);

    uh[id] = 0.0;
    vh[id] = 0.0;
  }

  //set boundaries
  //bottom
  if (i == 0 && j > 0 && j < nx + 1)
  {
    id = ID_2D(i, j, nx);
    id_boundary = ID_2D(i + 1, j, nx);

    h[id] = h[id_boundary];
    uh[id] = 0.0;
    vh[id] = 0.0;
  }

  //top
  if (i == ny + 1 && j > 0 && j < nx + 1)
  {
    id = ID_2D(i, j, nx);
    id_boundary = ID_2D(i - 1, j, nx);

    h[id] = h[id_boundary];
    uh[id] = 0.0;
    vh[id] = 0.0;
  }

  //left
  if ( j == 0 && i > 0 && i < ny + 1)
  {
    id = ID_2D(i, j, nx);
    id_boundary = ID_2D(i, j + 1, nx);

    h[id] = h[id_boundary];
    uh[id] = 0.0;
    vh[id] = 0.0;
  }

  //right
  if (j == nx + 1 && i > 0 && i < ny + 1)
  {
    id = ID_2D(i, j, nx);
    id_boundary = ID_2D(i, j - 1, nx);

    h[id] = h[id_boundary];
    uh[id] = 0.0;
    vh[id] = 0.0;
  }
  return;
}
/******************************************************************************/

__global__ void computeFluxesGPU(float *h, float *uh, float *vh, float *fh, float *fuh, float *fvh, float *gh, float *guh, float *gvh, int nx, int ny) 
{
  unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
  
  if (i >= nx + 2 || j >= ny + 2) // Bounds check
  return;
  
  unsigned int id = ID_2D(i, j, nx);

  float g = 9.81f; // Gravitational acceleration
  float h_safe = fmaxf(h[id], EPSILON); // Prevent division by zero
  
  // Compute fluxes safely
  fh[id] = uh[id];

  fuh[id] = uh[id] * uh[id] / h_safe + 0.5f * g * h_safe * h_safe;

  fvh[id] = uh[id] * vh[id] / h_safe;

  gh[id] = vh[id];

  guh[id] = uh[id] * vh[id] / h_safe;

  gvh[id] = vh[id] * vh[id] / h_safe + 0.5f * g * h_safe * h_safe;
  
}
/******************************************************************************/

__global__ void computeVariablesGPU(float *hm, float *uhm, float *vhm, float *fh, float *fuh, float *fvh, float *gh, float *guh, float *gvh, float *h, float *uh, float *vh, float lambda_x, float lambda_y, int nx, int ny)
{
  unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned int id, id_left, id_right, id_bottom, id_top;

  if (i > 0 && i < ny + 1 && j > 0 && j < nx + 1)  // Ensure proper bounds
  {
    id = ID_2D(i, j, nx);

    id_left   = ID_2D(i, j - 1, nx);
    id_right  = ID_2D(i, j + 1, nx);
    id_bottom = ID_2D(i - 1, j, nx);
    id_top    = ID_2D(i + 1, j, nx);

    hm[id] = 0.25 * (h[id_left] + h[id_right] + h[id_bottom] + h[id_top])
          - lambda_x * (fh[id_right] - fh[id_left])
          - lambda_y * (gh[id_top] - gh[id_bottom]);

    uhm[id] = 0.25 * (uh[id_left] + uh[id_right] + uh[id_bottom] + uh[id_top])
            - lambda_x * (fuh[id_right] - fuh[id_left])
            - lambda_y * (guh[id_top] - guh[id_bottom]);

    vhm[id] = 0.25 * (vh[id_left] + vh[id_right] + vh[id_bottom] + vh[id_top])
            - lambda_x * (fvh[id_right] - fvh[id_left])
            - lambda_y * (gvh[id_top] - gvh[id_bottom]);
  }
}
/******************************************************************************/

__global__ void updateVariablesGPU(float *h, float *uh, float *vh, float *hm, float *uhm, float *vhm, int nx, int ny)
{
  unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned int id;

  if (i > 0 && i < ny + 1 && j > 0 && j < nx + 1)  // Ensure proper bounds
  {
    id = ID_2D(i, j, nx);

    h[id] = hm[id];
    uh[id] = uhm[id];
    vh[id] = vhm[id];
  }
}
/******************************************************************************/

__global__ void applyBoundaryConditionsGPU(float *h, float *uh, float *vh, int nx, int ny, int bc_type)
{
  unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;

  unsigned int id, id_ghost;

  if (bc_type == 1) // Dirichlet Boundary Conditions
  {  
    // Left Boundary (j = 0)
    if (j == 0 && i >= 1 && i <= ny) 
    {
      id = ID_2D(i, 1, nx);
      id_ghost = ID_2D(i, 0, nx);
      h[id_ghost]  = h[id];
      uh[id_ghost] = uh[id];
      vh[id_ghost] = vh[id];
    }

    // Right Boundary (j = nx + 1)
    if (j == nx + 1 && i >= 1 && i <= ny) 
    {
      id = ID_2D(i, nx, nx);
      id_ghost = ID_2D(i, nx + 1, nx);
      h[id_ghost]  = h[id];
      uh[id_ghost] = uh[id];
      vh[id_ghost] = vh[id];
    }

    // Bottom Boundary (i = 0)
    if (i == 0 && j >= 1 && j <= nx) 
    {
      id = ID_2D(1, j, nx);
      id_ghost = ID_2D(0, j, nx);
      h[id_ghost]  = h[id];
      uh[id_ghost] = uh[id];
      vh[id_ghost] = vh[id];
    }

    // Top Boundary (i = ny + 1)
    if (i == ny + 1 && j >= 1 && j <= nx) 
    {
      id = ID_2D(ny, j, nx);
      id_ghost = ID_2D(ny + 1, j, nx);
      h[id_ghost]  = h[id];
      uh[id_ghost] = uh[id];
      vh[id_ghost] = vh[id];
    }
  }

  else if (bc_type == 2) // Periodic Boundary Conditions
  {  
    // Left to Right Periodic Boundary (wraps leftmost to rightmost)
    if (j == 0 && i >= 1 && i <= ny) 
    {
      id = ID_2D(i, nx, nx);
      id_ghost = ID_2D(i, 0, nx);
      h[id_ghost]  = h[id];
      uh[id_ghost] = uh[id];
      vh[id_ghost] = vh[id];
    }

    // Right to Left Periodic Boundary (wraps rightmost to leftmost)
    if (j == nx + 1 && i >= 1 && i <= ny) 
    {
      id = ID_2D(i, 1, nx);
      id_ghost = ID_2D(i, nx + 1, nx);
      h[id_ghost]  = h[id];
      uh[id_ghost] = uh[id];
      vh[id_ghost] = vh[id];
    }

    // Bottom to Top Periodic Boundary (wraps bottom to top)
    if (i == 0 && j >= 1 && j <= nx) 
    {
      id = ID_2D(ny, j, nx);
      id_ghost = ID_2D(0, j, nx);
      h[id_ghost]  = h[id];
      uh[id_ghost] = uh[id];
      vh[id_ghost] = vh[id];
    }

    // Top to Bottom Periodic Boundary (wraps top to bottom)
    if (i == ny + 1 && j >= 1 && j <= nx) 
    {
      id = ID_2D(1, j, nx);
      id_ghost = ID_2D(ny + 1, j, nx);
      h[id_ghost]  = h[id];
      uh[id_ghost] = uh[id];
      vh[id_ghost] = vh[id];
    }  
  }
  else if (bc_type == 3) // Reflective Boundary Conditions
  {  
    // Left Boundary (j = 1) - Reflective
    if (j == 1 && i > 0 && i < ny + 1) 
    {
      id = ID_2D(i, j, nx);
      id_ghost = ID_2D(i, j - 1, nx);
      h[id_ghost]  = h[id];
      uh[id_ghost] = -uh[id];  // Flip normal velocity
      vh[id_ghost] = vh[id];   // Keep tangential velocity
    }

    // Right Boundary (j = nx) - Reflective
    if (j == nx && i > 0 && i < ny + 1) 
    {
      id = ID_2D(i, j, nx);
      id_ghost = ID_2D(i, j + 1, nx);
      h[id_ghost]  = h[id];
      uh[id_ghost] = -uh[id];  // Flip normal velocity
      vh[id_ghost] = vh[id];   // Keep tangential velocity
    }

    // Bottom Boundary (i = 1) - Reflective
    if (i == 1 && j > 0 && j < nx + 1) 
    {
      id = ID_2D(i, j, nx);
      id_ghost = ID_2D(i - 1, j, nx);
      h[id_ghost]  = h[id];
      uh[id_ghost] = uh[id];   // Keep tangential velocity
      vh[id_ghost] = -vh[id];  // Flip normal velocity
    }

    // Top Boundary (i = ny) - Reflective
    if (i == ny && j > 0 && j < nx + 1) 
    {
      id = ID_2D(i, j, nx);
      id_ghost = ID_2D(i + 1, j, nx);
      h[id_ghost]  = h[id];
      uh[id_ghost] = uh[id];   // Keep tangential velocity
      vh[id_ghost] = -vh[id];  // Flip normal velocity
    }
  }
}
// ****************************************************************************************************************** //

// ****************************************************** MAIN ****************************************************** //
int main ( int argc, char *argv[] )
{ 
// ************************************************** INSTANTIATION ************************************************* //
  int k;
  int nx; 
  int ny; 

  float *x, *d_x;
  float *y, *d_y;

  float dx;
  float dy;
  double dt;
  float x_length;

  double time; 
  float t_final;

  // pointers to host, device memory 
  float *h, *d_h;
  float *uh, *d_uh;
  float *vh, *d_vh;

  float *fh, *d_fh;
  float *fuh, *d_fuh;
  float *fvh, *d_fvh;

  float *gh, *d_gh;
  float *guh, *d_guh;
  float *gvh, *d_gvh;

  float *hm, *d_hm; 
  float *uhm, *d_uhm;
  float *vhm, *d_vhm;

  // get command line arguments
  getArgs(&nx, &dt, &x_length, &t_final, argc, argv);
  ny = nx; // we assume this, does not have to be this way

  // Define the locations of the nodes and time steps and the spacing.
  dx = x_length / ( float ) ( nx );
  dy = x_length / ( float ) ( nx );

  float lambda_x = 0.5  * (float) dt / dx;
  float lambda_y = 0.5 * (float) dt / dy;

  // Define the block and grid sizes
  int dimx = 32;
  int dimy = 32;
  dim3 blockSize(dimx, dimy);
  dim3 gridSize((nx + 2 + blockSize.x - 1) / blockSize.x, (ny + 2 + blockSize.y - 1) / blockSize.y);

  // ************************************************ MEMORY ALLOCATIONS ************************************************ //

  // **** Allocate memory on host ****
  // Allocate space (nx+2)((nx+2) long, to account for ghosts
  // height array
  h  = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  hm = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  fh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  gh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );

  // x momentum array
  uh  = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  uhm = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  fuh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  guh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );

  // y momentum array
  vh  = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  vhm = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  fvh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  gvh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );

  // location arrays
  x = ( float * ) malloc ( nx * sizeof ( float ) );
  y = ( float * ) malloc ( ny * sizeof ( float ) );

  // **** Allocate memory on device ****

  CHECK(hipMalloc((void **)&d_x, nx * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_y, ny * sizeof ( float )));

  // Allocate space (nx+2)((nx+2) long, to account for ghosts
  CHECK(hipMalloc((void **)&d_h, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_uh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_vh, (nx+2)*(ny+2) * sizeof ( float )));

  CHECK(hipMalloc((void **)&d_fh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_fuh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_fvh, (nx+2)*(ny+2) * sizeof ( float )));

  CHECK(hipMalloc((void **)&d_gh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_guh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_gvh, (nx+2)*(ny+2) * sizeof ( float )));

  CHECK(hipMalloc((void **)&d_hm, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_uhm, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_vhm, (nx+2)*(ny+2) * sizeof ( float )));

  // ************************************************ INITIAL CONDITIONS ************************************************ //

  printf ( "\n" );
  printf ( "SHALLOW_WATER_2D\n" );
  printf ( "\n" );

  // set initial time & step counter
  // set time to zero and step counter to zero
  time = 0.0f;
  k = 0;

  // Apply the initial conditions.
  //initialConditionsGPU<<<gridSize, blockSize>>>(nx, ny, dx, dy, x_length, d_x, d_y, d_h, d_uh, d_vh);

  // Move data to the Host for initial conditions file write
  //CHECK(hipMemcpy(h, d_h, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyDeviceToHost));
  //CHECK(hipMemcpy(uh, d_uh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyDeviceToHost));
  //CHECK(hipMemcpy(vh, d_vh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyDeviceToHost));

  //CHECK(hipMemcpy(x, d_x, nx * sizeof ( float ), hipMemcpyDeviceToHost));
  //CHECK(hipMemcpy(y, d_y, nx * sizeof ( float ), hipMemcpyDeviceToHost));

  // **** INITIAL CONDITIONS ****
  //Apply the initial conditions.
  //printf("Before initial conditions\n");
  initial_conditions(nx, ny, dx, dy, x_length, x, y, h, uh, vh);

  // Write initial condition to a file
  writeResults(h, uh, vh, x, y, time, nx, ny);

  // Move data to the device for calculations
  CHECK(hipMemcpy(d_h, h, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_uh, uh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_vh, vh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyHostToDevice));

  // ******************************************************************** COMPUTATION SECTION ******************************************************************** //

  // start timer
  clock_t time_start = clock();

  while (time < t_final) // time loop begins
  {
    // Take a time step and increase step counter
    time = time + dt;
    k++;

    // **** COMPUTE FLUXES ****
    computeFluxesGPU<<<gridSize, blockSize>>>(d_h, d_uh, d_vh, d_fh, d_fuh, d_fvh, d_gh, d_guh, d_gvh, nx, ny);
    
    // **** COMPUTE VARIABLES ****
    computeVariablesGPU<<<gridSize, blockSize>>>(d_hm, d_uhm, d_vhm, d_fh, d_fuh, d_fvh, d_gh, d_guh, d_gvh, d_h, d_uh, d_vh, lambda_x, lambda_y, nx, ny);
  
    // **** UPDATE VARIABLES ****
    updateVariablesGPU<<<gridSize, blockSize>>>(d_h, d_uh, d_vh, d_hm, d_uhm, d_vhm, nx, ny);

    // **** APPLY BOUNDARY CONDITIONS ****
    applyBoundaryConditionsGPU<<<gridSize, blockSize>>>(d_h, d_uh, d_vh, nx, ny, 3);  

    CHECK(hipMemcpy(h, d_h, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(uh, d_uh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(vh, d_vh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyDeviceToHost));

    writeResults(h, uh, vh, x, y, time, nx, ny);

    CHECK(hipMemcpy(d_h, h, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_uh, uh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_vh, vh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyHostToDevice));

  } // end time loop

  // stop timer
  clock_t time_end = clock();
  double time_elapsed = (double)(time_end - time_start) / CLOCKS_PER_SEC;

  // Print out the results
  printf("Problem size: %d, time steps taken: %d,  elapsed time: %f s\n", nx, k, time_elapsed);

  // ******************************************************************** POSTPROCESSING ******************************************************************** //

  // Move data back to the host
  //CHECK(hipMemcpy(h, d_h, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyDeviceToHost));
  //CHECK(hipMemcpy(uh, d_uh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyDeviceToHost));
  //CHECK(hipMemcpy(vh, d_vh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyDeviceToHost));

  //writeResults(h, uh, vh, x, y, time, nx, ny);

  // ******************************************************************** DEALLOCATE MEMORY ******************************************************************** //

  //Free device memory.
  CHECK(hipFree(d_h));
  CHECK(hipFree(d_uh));
  CHECK(hipFree(d_vh));

  CHECK(hipFree(d_fh));
  CHECK(hipFree(d_fuh));
  CHECK(hipFree(d_fvh));

  CHECK(hipFree(d_gh));
  CHECK(hipFree(d_guh));
  CHECK(hipFree(d_gvh));

  CHECK(hipFree(d_hm));
  CHECK(hipFree(d_uhm));
  CHECK(hipFree(d_vhm));

  CHECK(hipFree(d_x));
  CHECK(hipFree(d_y));

  // Free host memory.
  free ( h );
  free ( uh );
  free ( vh ); 

  free ( fh );
  free ( fuh );
  free ( fvh );

  free ( gh );
  free ( guh );
  free ( gvh ); 

  free ( x );
  free ( y );

  // Terminate.
  printf ( "\n" );
  printf ( "SHALLOW_WATER_2D:\n" );
  printf ( "  Normal end of execution.\n" );
  printf ( "\n" );

  return 0;
}
// ******************************************************************************************************************************************** //

// ************************************************ SERIAL CODE ************************************************ //
// **** COMPUTE FLUXES ****
//Compute fluxes (including ghosts)
/* 
for ( i = 0; i < ny+2; i++ )
  for ( j = 0; j < nx+2; j++)
  {
    id = ID_2D(i,j,nx);

    fh[id] = uh[id]; //flux for the height equation: u*h
    fuh[id] = uh[id] * uh[id] / h[id] + 0.5 * g * h[id] * h[id]; //flux for the momentum equation: u^2*h + 0.5*g*h^2
    fvh[id] = uh[id] * vh[id] / h[id]; //flux for the momentum equation: u*v**h 
    gh[id] = vh[id]; //flux for the height equation: v*h
    guh[id] = uh[id] * vh[id] / h[id]; //flux for the momentum equation: u*v**h 
    gvh[id] = vh[id] * vh[id] / h[id] + 0.5 * g * h[id] * h[id]; //flux for the momentum equation: v^2*h + 0.5*g*h^2
  }

// **** COMPUTE VARIABLES ****
//Compute updated variables
for ( i = 1; i < ny + 1; i++ )
  for ( j = 1; j < nx + 1; j++ )
  {
    id=ID_2D(i,j,nx);
    id_left=ID_2D(i,j-1,nx);
    id_right=ID_2D(i,j+1,nx);
    id_bottom=ID_2D(i-1,j,nx);
    id_top=ID_2D(i+1,j,nx);

    hm[id] = 0.25*(h[id_left]+h[id_right]+h[id_bottom]+h[id_top]) 
      - lambda_x * ( fh[id_right] - fh[id_left] ) 
      - lambda_y * ( gh[id_top] - gh[id_bottom] );

    uhm[id] = 0.25*(uh[id_left]+uh[id_right]+uh[id_bottom]+uh[id_top]) 
      - lambda_x * ( fuh[id_right] - fuh[id_left] ) 
      - lambda_y * ( guh[id_top] - guh[id_bottom] );

    vhm[id] = 0.25*(vh[id_left]+vh[id_right]+vh[id_bottom]+vh[id_top]) 
      - lambda_x * ( fvh[id_right] - fvh[id_left] ) 
      - lambda_y * ( gvh[id_top] - gvh[id_bottom] );
  }

// **** UPDATE VARIABLES ****
//update interior state variables
for (i = 1; i < ny+1; i++)
  for (j = 1; j < nx+1; j++)
  {
  id=ID_2D(i,j,nx);
  h[id] = hm[id];
  uh[id] = uhm[id];
  vh[id] = vhm[id];
  }

// **** APPLY BOUNDARY CONDITIONS ****
//Update the ghosts (boundary conditions)

//left
j = 1;
for(i = 1; i < ny + 1; i++)
  {

    id = ID_2D(i, j, nx);

    id_left = ID_2D(i, j - 1, nx);

    h[id_left]  = h[id];

    uh[id_left] = - uh[id];

    vh[id_left] = vh[id];

  }

//right
j = nx;
for(i = 1; i < ny + 1; i++)
  {

    id = ID_2D(i, j, nx);

    id_right = ID_2D(i, j + 1, nx);

    h[id_right]  = h[id];

    uh[id_right] = - uh[id];

    vh[id_right] = vh[id];

  }

//bottom
i = 1;
for(j = 1; j < nx + 1; j++)
  {

    id = ID_2D(i, j, nx);

    id_bottom = ID_2D(i - 1, j, nx);

    h[id_bottom]  = h[id];

    uh[id_bottom] = uh[id];

    vh[id_bottom] = - vh[id];

  }

//top
i = ny;
for(j = 1; j < nx + 1; j++)
  {

    id = ID_2D(i, j, nx);

    id_top = ID_2D(i + 1, j, nx);

    h[id_top]  = h[id];

    uh[id_top] = uh[id];

    vh[id_top] = - vh[id];

  }
*/

/*
for ( i = 1; i < nx+1; i++ )
  {
    x[i-1] = -x_length/2+dx/2+(i-1)*dx;
    y[i-1] = -x_length/2+dy/2+(i-1)*dy;
  }

for ( i = 1; i < nx+1; i++ )
  for( j = 1; j < ny+1; j++)
  {
    float xx = x[j-1];
    float yy = y[i-1];
    id=ID_2D(i,j,nx);
    h[id] = 1.0 + 0.4*exp ( -5 * ( xx*xx + yy*yy) );
  } 
for ( i = 1; i < nx+1; i++ )
  for( j = 1; j < ny+1; j++)
  {
    id=ID_2D(i,j,nx);
    uh[id] = 0.0;
    vh[id] = 0.0;
  }
    
  i = 0
  for( j = 1; j < nx+1; j++)
  {
    id=ID_2D(i,j,nx);
    id1=ID_2D(i+1,j,nx);

    h[id] = h[id1];
    uh[id] = 0.0;
    vh[id] = 0.0;
  }

  i=nx+1;
  for( j = 1; j < nx+1; j++)
  {
    id=ID_2D(i,j,nx);
    id1=ID_2D(i-1,j,nx);

    h[id] = h[id1];
    uh[id] = 0.0;
    vh[id] = 0.0;
  } 
      
  j=0;
  for( i = 1; i < ny+1; i++)
  {
    id=ID_2D(i,j,nx);
    id1=ID_2D(i,j+1,nx);

    h[id] = h[id1];
    uh[id] = 0.0;
    vh[id] = 0.0;
  } 
  
  j=nx+1;
  for( i = 1; i < ny+1; i++)
  {
    id=ID_2D(i,j,nx);
    id1=ID_2D(i,j-1,nx);

    h[id] = h[id1];
    uh[id] = 0.0;
    vh[id] = 0.0;
  }

*/

/*
//Move data to the device for all GPU calculations
  CHECK(hipMemcpy(d_h, h, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_uh, uh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_vh, vh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyHostToDevice));

*/
