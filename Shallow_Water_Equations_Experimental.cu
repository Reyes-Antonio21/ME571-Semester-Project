# include <stdlib.h>
# include <stdio.h>
# include <math.h>
# include <string.h>
# include <time.h>
# include <chrono>
# include <hip/hip_runtime.h>

// ************************************************ UTILITIES ************************************************ //

void getArgs(int *nx, float *dt, float *x_length, float *finalRuntime, int argc, char *argv[])
{
  // Get the quadrature file root name:

  if ( argc <= 1 ){
    *nx = 400;
  }else{
    *nx = atoi ( argv[1] );
  }
  
  if ( argc <= 2 ){
    *dt = 0.002;
  }else{
    *dt = atof ( argv[2] );
  }
  
  if ( argc <= 3 ){
    *x_length = 10.0;
  }else{
    *x_length = atof ( argv[3] );
    }
  
  if ( argc <= 4 ){
    *finalRuntime = 0.5;
  }else{
    *finalRuntime = atof ( argv[4] );
  }
}
// ****************************************************************************** //

void writeResults(float h[], float uh[], float vh[], float x[], float y[], float time, int nx, int ny)
{
  char filename[50];

  int i, j, id;

  //Create the filename based on the time step.
  sprintf(filename, "tc2d_%08.6f.dat", time);

  //Open the file.
  FILE *file = fopen (filename, "wt" );
    
  if (!file)
  {
    fprintf (stderr, "\n" );

    fprintf (stderr, "WRITE_RESULTS - Fatal error!\n");

    fprintf (stderr, "  Could not open the output file.\n");

    exit (1);
  }

  else
  {  
    //Write the data.
    for ( i = 0; i < ny; i++ ) 
      for ( j = 0; j < nx; j++ )
      {
        id = ((i + 1)*(nx + 2)+(j + 1));
        fprintf ( file, "%24.16g\t%24.16g\t%24.16g\t %24.16g\t %24.16g\n", x[j], y[i], h[id], uh[id], vh[id]);
      }
    
    //Close the file.
    fclose (file);
  }

  return;
}
// ****************************************************************************** //

__global__ void initializeInterior(float *x, float *y, float *h, float *uh, float *vh, int nx, int ny, float dx, float dy, float x_length)
{
  unsigned int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
  unsigned int j = blockIdx.x * blockDim.x + threadIdx.x + 1;

  if (i < ny + 1 && j < nx + 1)
  {
    int id = i * (nx + 2) + j;

    float xx = -x_length / 2.0f + dx / 2.0f + (j - 1) * dx;
    float yy = -x_length / 2.0f + dy / 2.0f + (i - 1) * dy;

    x[j - 1] = xx;
    y[i - 1] = yy;

    h[id] += 1.0f + 0.40f * expf(-5.0f * (xx * xx + yy * yy));
  }
}
// ****************************************************************************** //

__global__ void applyLeftBoundary(float *h, float *uh, float *vh, int nx, int ny)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i > 0 && i < ny + 1)
  {
    int id = i * (nx + 2);
    int id_interior = i * (nx + 2) + 1;

    float h_val = h[id_interior];
    float uh_val = uh[id_interior];
    float vh_val = vh[id_interior];

    h[id]  = h_val;
    uh[id] = -uh_val;
    vh[id] =  vh_val;
  }
}
// ****************************************************************************** //

__global__ void applyRightBoundary(float *h, float *uh, float *vh, int nx, int ny)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i > 0 && i < ny + 1)
  {
    int nx_ext = nx + 2;
    int id = i * nx_ext + (nx + 1);
    int id_interior = i * nx_ext + nx;

    float h_val = h[id_interior];
    float uh_val = uh[id_interior];
    float vh_val = vh[id_interior];

    h[id]  = h_val;
    uh[id] = -uh_val;
    vh[id] =  vh_val;
  }
}
// ****************************************************************************** //

__global__ void applyBottomBoundary(float *h, float *uh, float *vh, int nx, int ny)
{
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  if (j > 0 && j < nx + 1)
  {
    int nx_ext = nx + 2;
    int id = j;
    int id_interior = 1 * nx_ext + j;

    float h_val = h[id_interior];
    float uh_val = uh[id_interior];
    float vh_val = vh[id_interior];

    h[id]  = h_val;
    uh[id] =  uh_val;
    vh[id] = -vh_val;
  }
}
// ****************************************************************************** //

__global__ void applyTopBoundary(float *h, float *uh, float *vh, int nx, int ny)
{
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  if (j > 0 && j < nx + 1)
  {
    int nx_ext = nx + 2;
    int id = (ny + 1) * nx_ext + j;
    int id_interior = ny * nx_ext + j;

    float h_val = h[id_interior];
    float uh_val = uh[id_interior];
    float vh_val = vh[id_interior];

    h[id]  = h_val;
    uh[id] =  uh_val;
    vh[id] = -vh_val;
  }
}
// ****************************************************************************** //

__device__ void haloExchange(float* sh_h, float* sh_uh, float* sh_vh, const float* h, const float* uh, const float* vh, int i, int j, int local_i, int local_j, int nx, int ny, int blockDim_x)
{
  #define SH_ID(i, j, blockDim_x) ((i) * (blockDim_x + 2) + (j))
  #define ID_2D(i, j, nx) ((i) * (nx + 2) + (j))

  int global_id, halo_global_id;
  int local_halo_id;

  // === LEFT Halo ===
  if (threadIdx.x == 0 && j > 0)
  {
    halo_global_id = ID_2D(i, j - 1, nx);
    local_halo_id  = SH_ID(local_i, local_j - 1, blockDim_x);

    sh_h[local_halo_id]  = h[halo_global_id];
    sh_uh[local_halo_id] = uh[halo_global_id];
    sh_vh[local_halo_id] = vh[halo_global_id];
  }
  else if (threadIdx.x == 0 && j == 0)
  {
    global_id = ID_2D(i, j, nx);
    local_halo_id = SH_ID(local_i, local_j - 1, blockDim_x);

    sh_h[local_halo_id]  = h[global_id];
    sh_uh[local_halo_id] = -uh[global_id];
    sh_vh[local_halo_id] =  vh[global_id];
  }

  // === RIGHT Halo ===
  if (threadIdx.x == blockDim.x - 1 && j < nx + 1)
  {
    halo_global_id = ID_2D(i, j + 1, nx);
    local_halo_id  = SH_ID(local_i, local_j + 1, blockDim_x);

    sh_h[local_halo_id]  = h[halo_global_id];
    sh_uh[local_halo_id] = uh[halo_global_id];
    sh_vh[local_halo_id] = vh[halo_global_id];
  }
  else if (threadIdx.x == blockDim.x - 1 && j == nx + 1)
  {
    global_id = ID_2D(i, j, nx);
    local_halo_id = SH_ID(local_i, local_j + 1, blockDim_x);

    sh_h[local_halo_id]  = h[global_id];
    sh_uh[local_halo_id] = -uh[global_id];
    sh_vh[local_halo_id] =  vh[global_id];
  }

  // === BOTTOM Halo ===
  if (threadIdx.y == 0 && i > 0)
  {
    halo_global_id = ID_2D(i - 1, j, nx);
    local_halo_id  = SH_ID(local_i - 1, local_j, blockDim_x);

    sh_h[local_halo_id]  = h[halo_global_id];
    sh_uh[local_halo_id] = uh[halo_global_id];
    sh_vh[local_halo_id] = vh[halo_global_id];
  }
  else if (threadIdx.y == 0 && i == 0)
  {
    global_id = ID_2D(i, j, nx);
    local_halo_id = SH_ID(local_i - 1, local_j, blockDim_x);

    sh_h[local_halo_id]  = h[global_id];
    sh_uh[local_halo_id] =  uh[global_id];
    sh_vh[local_halo_id] = -vh[global_id];
  }

  // === TOP Halo ===
  if (threadIdx.y == blockDim.y - 1 && i < ny + 1)
  {
    halo_global_id = ID_2D(i + 1, j, nx);
    local_halo_id  = SH_ID(local_i + 1, local_j, blockDim_x);

    sh_h[local_halo_id]  = h[halo_global_id];
    sh_uh[local_halo_id] = uh[halo_global_id];
    sh_vh[local_halo_id] = vh[halo_global_id];
  }
  else if (threadIdx.y == blockDim.y - 1 && i == ny + 1)
  {
    global_id = ID_2D(i, j, nx);
    local_halo_id = SH_ID(local_i + 1, local_j, blockDim_x);

    sh_h[local_halo_id]  = h[global_id];
    sh_uh[local_halo_id] =  uh[global_id];
    sh_vh[local_halo_id] = -vh[global_id];
  }

  #undef ID_2D
  #undef SH_ID
}


__global__ void shallowWaterSolver(float *__restrict__ h, float *__restrict__ uh, float *__restrict__ vh, float lambda_x, float lambda_y, int nx, int ny, float dt, float finalRuntime)
{
  unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

  unsigned int local_i = threadIdx.y + 1;
  unsigned int local_j = threadIdx.x + 1;

  unsigned int id;
  unsigned int local_id, local_id_left, local_id_right, local_id_bottom, local_id_top;

  extern __shared__ float sharedmemory[];

  float *sh_h   = sharedmemory;
  float *sh_uh  = sh_h   + (blockDim.y + 2) * (blockDim.x + 2);
  float *sh_vh  = sh_uh  + (blockDim.y + 2) * (blockDim.x + 2);

  float *sh_fh  = sh_vh  + (blockDim.y + 2) * (blockDim.x + 2);
  float *sh_gh  = sh_fh  + (blockDim.y + 2) * (blockDim.x + 2);
  float *sh_fuh = sh_gh  + (blockDim.y + 2) * (blockDim.x + 2);
  float *sh_guh = sh_fuh + (blockDim.y + 2) * (blockDim.x + 2);
  float *sh_fvh = sh_guh + (blockDim.y + 2) * (blockDim.x + 2);
  float *sh_gvh = sh_fvh + (blockDim.y + 2) * (blockDim.x + 2);

  # define SH_ID(i, j, blockDim_x) ((i) * (blockDim_x + 2) + (j))
  # define ID_2D(i, j, nx) ((i) * (nx + 2) + (j))

  if (i > 0 && i < ny + 1 && j > 0 && j < nx + 1)
  {
    id = ID_2D(i, j, nx);
    local_id = SH_ID(local_i, local_j, blockDim.x);

    sh_h[local_id]  = h[id];
    sh_uh[local_id] = uh[id];
    sh_vh[local_id] = vh[id];
  }

  haloExchange(sh_h, sh_uh, sh_vh, h, uh, vh, i, j, local_i, local_j, nx, ny, blockDim.x);

  __syncthreads();

  float programRuntime = 0.0f;
  float g = 9.81f;
  float g_half = 0.5f * g;

  while (programRuntime < finalRuntime)
  {
    haloExchange(sh_h, sh_uh, sh_vh, h, uh, vh, i, j, local_i, local_j, nx, ny, blockDim.x);

    __syncthreads();

    if (i > 0 && i < ny + 1 && j > 0 && j < nx + 1)
    {
      local_id = SH_ID(local_i, local_j, blockDim.x);

      float h_val  = sh_h[local_id];
      float uh_val = sh_uh[local_id];
      float vh_val = sh_vh[local_id];

      float inv_h = 1.0f / h_val;
      float h2 = h_val * h_val;

      sh_fh[local_id] = uh_val;
      sh_gh[local_id] = vh_val;

      float uh2 = uh_val * uh_val;
      float vh2 = vh_val * vh_val;
      float uv = uh_val * vh_val;

      sh_fuh[local_id] = __fmaf_rn(uh2, inv_h, g_half * h2);
      sh_fvh[local_id] = uv * inv_h;           

      sh_guh[local_id] = uv * inv_h;           
      sh_gvh[local_id] = __fmaf_rn(vh2, inv_h, g_half * h2);
    }

    __syncthreads();

    if (i > 0 && i < ny + 1 && j > 0 && j < nx + 1)
    {
      local_id       = SH_ID(local_i, local_j, blockDim.x);
      local_id_left  = SH_ID(local_i, local_j - 1, blockDim.x);
      local_id_right = SH_ID(local_i, local_j + 1, blockDim.x);
      local_id_bottom = SH_ID(local_i - 1, local_j, blockDim.x);
      local_id_top    = SH_ID(local_i + 1, local_j, blockDim.x);

      float h_l  = sh_h[local_id_left];
      float h_r  = sh_h[local_id_right];
      float h_b  = sh_h[local_id_bottom];
      float h_t  = sh_h[local_id_top];

      float uh_l = sh_uh[local_id_left];
      float uh_r = sh_uh[local_id_right];
      float uh_b = sh_uh[local_id_bottom];
      float uh_t = sh_uh[local_id_top];

      float vh_l = sh_vh[local_id_left];
      float vh_r = sh_vh[local_id_right];
      float vh_b = sh_vh[local_id_bottom];
      float vh_t = sh_vh[local_id_top];

      float fh_l = sh_fh[local_id_left];
      float fh_r = sh_fh[local_id_right];
      float gh_b = sh_gh[local_id_bottom];
      float gh_t = sh_gh[local_id_top];

      float fuh_l = sh_fuh[local_id_left];
      float fuh_r = sh_fuh[local_id_right];
      float guh_b = sh_guh[local_id_bottom];
      float guh_t = sh_guh[local_id_top];

      float fvh_l = sh_fvh[local_id_left];
      float fvh_r = sh_fvh[local_id_right];
      float gvh_b = sh_gvh[local_id_bottom];
      float gvh_t = sh_gvh[local_id_top];

      sh_h[local_id] = __fmaf_rn(-lambda_x, (fh_r - fh_l), __fmaf_rn(-lambda_y, (gh_t - gh_b), 0.25f * (h_l + h_r + h_b + h_t)));
      sh_uh[local_id] = __fmaf_rn(-lambda_x, (fuh_r - fuh_l), __fmaf_rn(-lambda_y, (guh_t - guh_b), 0.25f * (uh_l + uh_r + uh_b + uh_t)));
      sh_vh[local_id] = __fmaf_rn(-lambda_x, (fvh_r - fvh_l), __fmaf_rn(-lambda_y, (gvh_t - gvh_b), 0.25f * (vh_l + vh_r + vh_b + vh_t)));
    }

    __syncthreads();

    // Correct boundary condition application using shared memory coordinates
    if (local_j == 0 && local_i > 0 && local_i < blockDim.y + 1)
    {
      sh_h[SH_ID(local_i, local_j, blockDim.x)] = sh_h[SH_ID(local_i, local_j + 1, blockDim.x)];
      sh_uh[SH_ID(local_i, local_j, blockDim.x)] = -sh_uh[SH_ID(local_i, local_j + 1, blockDim.x)];
      sh_vh[SH_ID(local_i, local_j, blockDim.x)] = sh_vh[SH_ID(local_i, local_j + 1, blockDim.x)];
    }

    if (local_j == blockDim.x + 1 && local_i > 0 && local_i < blockDim.y + 1)
    {
      sh_h[SH_ID(local_i, local_j, blockDim.x)] = sh_h[SH_ID(local_i, local_j - 1, blockDim.x)];
      sh_uh[SH_ID(local_i, local_j, blockDim.x)] = -sh_uh[SH_ID(local_i, local_j - 1, blockDim.x)];
      sh_vh[SH_ID(local_i, local_j, blockDim.x)] = sh_vh[SH_ID(local_i, local_j - 1, blockDim.x)];
    }

    if (local_i == 0 && local_j > 0 && local_j < blockDim.x + 1)
    {
      sh_h[SH_ID(local_i, local_j, blockDim.x)] = sh_h[SH_ID(local_i + 1, local_j, blockDim.x)];
      sh_uh[SH_ID(local_i, local_j, blockDim.x)] = sh_uh[SH_ID(local_i + 1, local_j, blockDim.x)];
      sh_vh[SH_ID(local_i, local_j, blockDim.x)] = -sh_vh[SH_ID(local_i + 1, local_j, blockDim.x)];
    }

    if (local_i == blockDim.y + 1 && local_j > 0 && local_j < blockDim.x + 1)
    {
      sh_h[SH_ID(local_i, local_j, blockDim.x)] = sh_h[SH_ID(local_i - 1, local_j, blockDim.x)];
      sh_uh[SH_ID(local_i, local_j, blockDim.x)] = sh_uh[SH_ID(local_i - 1, local_j, blockDim.x)];
      sh_vh[SH_ID(local_i, local_j, blockDim.x)] = -sh_vh[SH_ID(local_i - 1, local_j, blockDim.x)];
    }

    __syncthreads();

    programRuntime += dt;
  }

  // Final write-back to global memory
  if (i > 0 && i < ny + 1 && j > 0 && j < nx + 1)
  {
    id = ID_2D(i, j, nx);
    local_id = SH_ID(local_i, local_j, blockDim.x);

    h[id]  = sh_h[local_id];
    uh[id] = sh_uh[local_id];
    vh[id] = sh_vh[local_id];
  }
}
// ****************************************************************************************************************** //

// ****************************************************** MAIN ****************************************************** //
int main ( int argc, char *argv[] )
{ 
  // ************************************************** INSTANTIATION ************************************************* //
  int k;

  int nx; 
  int ny; 

  float dx;
  float dy;
  
  float x_length;

  float dt;
  float finalRuntime;
  
  // pointers to host, device memory 
  float *h, *d_h;
  float *uh, *d_uh;
  float *vh, *d_vh;

  float *fh, *d_fh;
  float *fuh, *d_fuh;
  float *fvh, *d_fvh;

  float *gh, *d_gh;
  float *guh, *d_guh;
  float *gvh, *d_gvh;

  float *hm, *d_hm; 
  float *uhm, *d_uhm;
  float *vhm, *d_vhm;

  float *x, *d_x;
  float *y, *d_y;

  // get command line arguments
  getArgs(&nx, &dt, &x_length, &finalRuntime, argc, argv);
  ny = nx; // we assume the grid is square

  // Define the locations of the nodes and time steps and the spacing.
  dx = x_length / ( float ) ( nx );
  dy = x_length / ( float ) ( nx );

  float lambda_x = 0.5f * dt / dx;
  float lambda_y = 0.5f * dt / dy;

  // Define the block and grid sizes
  int dimx = 32;
  int dimy = 32;
  dim3 blockSize(dimx, dimy);
  dim3 gridSize((nx + 2 + blockSize.x - 1) / blockSize.x, (ny + 2 + blockSize.y - 1) / blockSize.y);

  // Calculate shared memory size
  size_t sharedMemSize = ((10 * (blockSize.x+2) * (blockSize.y+2) * sizeof(float)) + 127) & ~127;

  hipFuncSetAttribute(reinterpret_cast<const void*>(shallowWaterSolver), hipFuncAttributeMaxDynamicSharedMemorySize, 98304);

  int boundaryBlockSize = 1024;
  int gridSizeY = (ny + boundaryBlockSize - 1) / boundaryBlockSize; 
  int gridSizeX = (nx + boundaryBlockSize - 1) / boundaryBlockSize;  

  // ************************************************ MEMORY ALLOCATIONS ************************************************ //

  // **** Allocate memory on host ****
  // Allocate space (nx+2)((nx+2) long, to account for ghosts
  // height array
  h  = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  hm = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  fh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  gh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );

  // x momentum array
  uh  = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  uhm = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  fuh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  guh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );

  // y momentum array
  vh  = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  vhm = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  fvh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  gvh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );

  // location arrays
  x = ( float * ) malloc ( nx * sizeof ( float ) );
  y = ( float * ) malloc ( ny * sizeof ( float ) );

  // **** Allocate memory on device ****

  // Allocate space (nx+2)((nx+2) long, to account for ghosts
  hipMalloc((void **)&d_h, (nx+2) * (ny+2) * sizeof ( float ));
  hipMalloc((void **)&d_uh, (nx+2) * (ny+2) * sizeof ( float ));
  hipMalloc((void **)&d_vh, (nx+2) * (ny+2) * sizeof ( float ));

  hipMalloc((void **)&d_fh, (nx+2) * (ny+2) * sizeof ( float ));
  hipMalloc((void **)&d_fuh, (nx+2) * (ny+2) * sizeof ( float ));
  hipMalloc((void **)&d_fvh, (nx+2) * (ny+2) * sizeof ( float ));

  hipMalloc((void **)&d_gh, (nx+2) * (ny+2) * sizeof ( float ));
  hipMalloc((void **)&d_guh, (nx+2) * (ny+2) * sizeof ( float ));
  hipMalloc((void **)&d_gvh, (nx+2) * (ny+2) * sizeof ( float ));

  hipMalloc((void **)&d_hm, (nx+2) * (ny+2) * sizeof ( float ));
  hipMalloc((void **)&d_uhm, (nx+2) * (ny+2) * sizeof ( float ));
  hipMalloc((void **)&d_vhm, (nx+2) * (ny+2) * sizeof ( float ));

  hipMalloc((void **)&d_x, nx * sizeof ( float ));
  hipMalloc((void **)&d_y, ny * sizeof ( float ));

  hipMemset(d_h, 0, (nx+2) * (ny+2) * sizeof ( float ));
  hipMemset(d_uh, 0, (nx+2) * (ny+2) * sizeof ( float ));
  hipMemset(d_vh, 0, (nx+2) * (ny+2) * sizeof ( float ));
  
  // *********************************************************************** INITIAL CONDITIONS ********************************************************************** //

  printf ( "\n" );
  printf ( "SHALLOW_WATER_2D\n" );
  printf ( "\n" );
  
  for(k = 1; k < 6; k++)
  {
    // Apply the initial conditions.
    initializeInterior<<<gridSize, blockSize>>>(d_x, d_y, d_h, d_uh, d_vh, nx, ny, dx, dy, x_length);

    applyLeftBoundary<<<gridSizeY, boundaryBlockSize>>>(d_h, d_uh, d_vh, nx, ny);

    applyRightBoundary<<<gridSizeY, boundaryBlockSize>>>(d_h, d_uh, d_vh, nx, ny);

    applyBottomBoundary<<<gridSizeX, boundaryBlockSize>>>(d_h, d_uh, d_vh, nx, ny);

    applyTopBoundary<<<gridSizeX, boundaryBlockSize>>>(d_h, d_uh, d_vh, nx, ny);

    if(k == 1 && nx == 200)
    {
      hipMemcpy(h, d_h, (nx+2) * (ny+2) * sizeof ( float ), hipMemcpyDeviceToHost);
      hipMemcpy(uh, d_uh, (nx+2) * (ny+2) * sizeof ( float ), hipMemcpyDeviceToHost);
      hipMemcpy(vh, d_vh, (nx+2) * (ny+2) * sizeof ( float ), hipMemcpyDeviceToHost);

      hipMemcpy(x, d_x, nx * sizeof ( float ), hipMemcpyDeviceToHost);
      hipMemcpy(y, d_y, ny * sizeof ( float ), hipMemcpyDeviceToHost);

      // Write initial condition to a file
      writeResults(h, uh, vh, x, y, 0.000000, nx, ny);
    }

    // ******************************************************************** COMPUTATION SECTION ******************************************************************** //

    // start program timer
    auto start_time = std::chrono::steady_clock::now();

    shallowWaterSolver<<<gridSize, blockSize, sharedMemSize>>>(d_h, d_uh, d_vh, lambda_x, lambda_y, nx, ny, dt, finalRuntime);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
    {
      printf("CUDA Error launching shallowWaterSolver: %s\n", hipGetErrorString(err));
    }

    hipDeviceSynchronize();  // Wait for kernel to finish

    // stop timer
    auto end_time = std::chrono::steady_clock::now();
    std::chrono::duration<double> time_elapsed = end_time - start_time;

    // Print out the results
    printf("Problem size: %d, Iteration: %d, Elapsed time: %f s\n", nx, k, time_elapsed);

    if(k == 1 && nx == 200)
    {
      hipMemcpy(h, d_h, (nx+2) * (ny+2) * sizeof ( float ), hipMemcpyDeviceToHost);
      hipMemcpy(uh, d_uh, (nx+2) * (ny+2) * sizeof ( float ), hipMemcpyDeviceToHost);
      hipMemcpy(vh, d_vh, (nx+2) * (ny+2) * sizeof ( float ), hipMemcpyDeviceToHost);

      // Write initial condition to a file
      writeResults(h, uh, vh, x, y, 0.500000, nx, ny);
    }
  }

  // ******************************************************************** DEALLOCATE MEMORY ******************************************************************** //

  //Free device memory.
  hipFree(d_h);
  hipFree(d_uh);
  hipFree(d_vh);

  hipFree(d_fh);
  hipFree(d_fuh);
  hipFree(d_fvh);

  hipFree(d_gh);
  hipFree(d_guh);
  hipFree(d_gvh);

  hipFree(d_hm);
  hipFree(d_uhm);
  hipFree(d_vhm);

  hipFree(d_x);
  hipFree(d_y);

  // Free host memory.
  free ( h );
  free ( uh );
  free ( vh ); 

  free ( fh );
  free ( fuh );
  free ( fvh );

  free ( gh );
  free ( guh );
  free ( gvh ); 

  free ( x );
  free ( y );

  // Terminate.
  printf ( "\n" );
  printf ( "SHALLOW_WATER_2D:\n" );
  printf ( "Normal end of execution.\n" );
  printf ( "\n" );

  return 0;
}
// ******************************************************************************************************************************************** //