# include <stdlib.h>
# include <stdio.h>
# include <math.h>
# include <string.h>
# include <time.h>
# include <chrono>
# include <iostream>
# include <hip/hip_runtime.h>

// ************************************************ UTILITIES ************************************************ //

void getArgs(int *nx, float *dt, float *x_length, float *finalRuntime, int argc, char *argv[])
{
  // Get the quadrature file root name:

  if ( argc <= 1 ){
    *nx = 400;
  }else{
    *nx = atoi ( argv[1] );
  }
  
  if ( argc <= 2 ){
    *dt = 0.002;
  }else{
    *dt = atof ( argv[2] );
  }
  
  if ( argc <= 3 ){
    *x_length = 10.0;
  }else{
    *x_length = atof ( argv[3] );
    }
  
  if ( argc <= 4 ){
    *finalRuntime = 0.5;
  }else{
    *finalRuntime = atof ( argv[4] );
  }
}
// ****************************************************************************** //

void writeResults(float h[], float uh[], float vh[], float x[], float y[], float time, int nx, int ny)
{
  char filename[50];

  int i, j, id;

  //Create the filename based on the time step.
  sprintf(filename, "tc2d_%08.6f.dat", time);

  //Open the file.
  FILE *file = fopen (filename, "wt" );
    
  if (!file)
  {
    fprintf (stderr, "\n" );

    fprintf (stderr, "WRITE_RESULTS - Fatal error!\n");

    fprintf (stderr, "  Could not open the output file.\n");

    exit (1);
  }

  else
  {  
    //Write the data.
    for ( i = 0; i < ny; i++ ) 
      for ( j = 0; j < nx; j++ )
      {
        id = ((i + 1)*(nx + 2)+(j + 1));
        fprintf ( file, "%24.16g\t%24.16g\t%24.16g\t %24.16g\t %24.16g\n", x[j], y[i], h[id], uh[id], vh[id]);
      }
    
    //Close the file.
    fclose (file);
  }

  return;
}
// ****************************************************************************** //

__global__ void initializeInterior(float *x, float *y, float *h, float *uh, float *vh, int nx, int ny, float dx, float dy, float x_length)
{
  unsigned int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
  unsigned int j = blockIdx.x * blockDim.x + threadIdx.x + 1;

  if (i < ny + 1 && j < nx + 1)
  {
    int id = i * (nx + 2) + j;

    float xx = -x_length / 2.0f + dx / 2.0f + (j - 1) * dx;
    float yy = -x_length / 2.0f + dy / 2.0f + (i - 1) * dy;

    x[j - 1] = xx;
    y[i - 1] = yy;

    h[id] += 1.0f + 0.40f * expf(-5.0f * (xx * xx + yy * yy));
  }
}
// ****************************************************************************** //

__global__ void applyLeftBoundary(float *h, float *uh, float *vh, int nx, int ny)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i > 0 && i < ny + 1)
  {
    int id = i * (nx + 2);
    int id_interior = i * (nx + 2) + 1;

    float h_val = h[id_interior];
    float uh_val = uh[id_interior];
    float vh_val = vh[id_interior];

    h[id]  = h_val;
    uh[id] = -uh_val;
    vh[id] =  vh_val;
  }
}
// ****************************************************************************** //

__global__ void applyRightBoundary(float *h, float *uh, float *vh, int nx, int ny)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i > 0 && i < ny + 1)
  {
    int id = i * (nx + 2) + (nx + 1);
    int id_interior = i * (nx + 2) + nx;

    float h_val = h[id_interior];
    float uh_val = uh[id_interior];
    float vh_val = vh[id_interior];

    h[id]  = h_val;
    uh[id] = -uh_val;
    vh[id] =  vh_val;
  }
}
// ****************************************************************************** //

__global__ void applyBottomBoundary(float *h, float *uh, float *vh, int nx, int ny)
{
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  if (j > 0 && j < nx + 1)
  {
    int id = j;
    int id_interior = 1 * (nx + 2) + j;

    float h_val = h[id_interior];
    float uh_val = uh[id_interior];
    float vh_val = vh[id_interior];

    h[id]  = h_val;
    uh[id] =  uh_val;
    vh[id] = -vh_val;
  }
}
// ****************************************************************************** //

__global__ void applyTopBoundary(float *h, float *uh, float *vh, int nx, int ny)
{
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  if (j > 0 && j < nx + 1)
  {
    int id = (ny + 1) * (nx + 2) + j;
    int id_interior = ny * (nx + 2) + j;

    float h_val = h[id_interior];
    float uh_val = uh[id_interior];
    float vh_val = vh[id_interior];

    h[id]  = h_val;
    uh[id] =  uh_val;
    vh[id] = -vh_val;
  }
}
// ****************************************************************************** //

__device__ void haloExchange(float* sh_h, float* sh_uh, float* sh_vh, const float* h, const float* uh, const float* vh, int global_i, int global_j, int local_i, int local_j, int nx, int ny, int blockDim_x, int blockDim_y)
{
  #define SH_ID(local_i, local_j) ((local_i) * (blockDim.x + 2) + (local_j)) 
  #define ID_2D(global_i, global_j) ((global_i) * (nx + 2) + (global_j))

  // === LEFT ===
  if (local_j == 1)
  {
    int global_id = ID_2D(0, global_j - 1);
    int local_id = SH_ID(local_i, local_j - 1);

    if (global_id >= 0) 
    {
      int global_id = ID_2D(global_i, global_j - 1);

      sh_h[local_id]  = h[global_id];
      sh_uh[local_id] = uh[global_id];
      sh_vh[local_id] = vh[global_id];
    }
  }

  // === RIGHT ===
  if (local_j == blockDim_x) 
  {
    int global_id = ID_2D(0, global_j + 1);
    int local_id = SH_ID(local_i, local_j + 1);

    if (global_id < nx + 2) 
    {
      int global_id = ID_2D(global_i, global_j + 1);

      sh_h[local_id]  = h[global_id];
      sh_uh[local_id] = uh[global_id];
      sh_vh[local_id] = vh[global_id];
    }
  }

  // === BOTTOM ===
  if (local_i == 1) 
  {
    int global_id = ID_2D(global_i - 1, 0);
    int local_id = SH_ID(local_i - 1, local_j);

    if (global_id >= 0) 
    {
      int global_id = ID_2D(global_i - 1, global_j);

      sh_h[local_id]  = h[global_id];
      sh_uh[local_id] = uh[global_id];
      sh_vh[local_id] = vh[global_id];
    }
  }

  // === TOP ===
  if (local_i == blockDim_y) 
  {
    int global_id = ID_2D(global_i + 1, 0);
    int local_id = SH_ID(local_i + 1, local_j);

    if (global_id < ny + 2) 
    { 
      int global_id = ID_2D(global_i + 1, global_j);
      sh_h[local_id]  = h[global_id];
      sh_uh[local_id] = uh[global_id];
      sh_vh[local_id] = vh[global_id];
    }
  }

  #undef ID_2D
  #undef SH_ID
}
// ****************************************************************************** //

__device__ void applyReflectiveBCs(float* sh_h, float* sh_uh, float* sh_vh, int local_i, int local_j, int blockDim_x, int blockDim_y)
{
  #define SH_ID(local_i, local_j) ((local_i) * (blockDim.x + 2) + (local_j))

  // Left boundary: reflect uh
  if (local_j == 1) 
  {
    int left_id = SH_ID(local_i, 0);
    int interior_id = SH_ID(local_i, 1);

    sh_h[left_id] = sh_h[interior_id];
    sh_uh[left_id] = -sh_uh[interior_id];
    sh_vh[left_id] =  sh_vh[interior_id];
  }

  // Right boundary: reflect uh
  if (local_j == blockDim_x) 
  {
    int right_id = SH_ID(local_i, blockDim_x + 1);
    int interior_id = SH_ID(local_i, blockDim_x);

    sh_h[right_id] = sh_h[interior_id];
    sh_uh[right_id] = -sh_uh[interior_id];
    sh_vh[right_id] =  sh_vh[interior_id];
  }

  // Bottom boundary: reflect vh
  if (local_i == 1) 
  {
    int bottom_id = SH_ID(0, local_j);
    int interior_id = SH_ID(1, local_j);

    sh_h[bottom_id] = sh_h[interior_id];
    sh_uh[bottom_id] = sh_uh[interior_id];
    sh_vh[bottom_id] = -sh_vh[interior_id];
  }

  // Top boundary: reflect vh
  if (local_i == blockDim_y) 
  {
    int top_id = SH_ID(blockDim_y + 1, local_j);
    int interior_id = SH_ID(blockDim_y, local_j);

    sh_h[top_id] = sh_h[interior_id];
    sh_uh[top_id] = sh_uh[interior_id];
    sh_vh[top_id] = -sh_vh[interior_id];
  }

  #undef SH_ID
}
// ****************************************************************************************************************** //

__device__ void writeGlobalToInterior(const float* d_mem, float* sh_mem, int global_i, int global_j, int local_i, int local_j, int nx, int ny, int blockDim_x, int blockDim_y)
{
  #define SH_ID(local_i, local_j) ((local_i) * (blockDim.x + 2) + (local_j)) 
  #define ID_2D(global_i, global_j) ((global_i) * (nx + 2) + (global_j))

  if (local_i > 0 && local_i < blockDim_y - 1 && local_j > 0 && local_j < blockDim_x - 1)
  {
    if (global_i > 0 && global_i < ny + 1 && global_j > 0 && global_j < nx + 1)
    {
      int global_id = ID_2D(global_i, global_j);
      int local_id = SH_ID(local_i, local_j);

      sh_mem[local_id] = d_mem[global_id];
    }
  }

  #undef SH_ID
  #undef ID_2D
}
// ****************************************************************************************************************** //

__device__ void writeInteriorToGlobal(float* d_mem, const float* sh_mem, int global_i, int global_j, int local_i, int local_j, int nx, int ny, int blockDim_x, int blockDim_y)
{
  #define SH_ID(local_i, local_j) ((local_i) * (blockDim.x + 2) + (local_j)) 
  #define ID_2D(global_i, global_j) ((global_i) * (nx + 2) + (global_j))

  if (global_i > 0 && global_i < ny + 1 && global_j > 0 && global_j < nx + 1)
  {
    if (local_i > 0 && local_i < blockDim_y - 1 && local_j > 0 && local_j < blockDim_x - 1)
    {
      int global_id = ID_2D(global_i, global_j);
      int local_id = SH_ID(local_i, local_j);

      d_mem[global_id] = sh_mem[local_id];
    }
  }
  
  #undef SH_ID
  #undef ID_2D
}
// ****************************************************************************************************************** //

__global__ void shallowWaterSolver(float *__restrict__ h, float *__restrict__ uh, float *__restrict__ vh, float lambda_x, float lambda_y, int nx, int ny, float dt, float finalRuntime)
{
  unsigned int global_i = blockIdx.y * blockDim.y + threadIdx.y + 1;
  unsigned int global_j = blockIdx.x * blockDim.x + threadIdx.x + 1;

  unsigned int local_i = threadIdx.y + 1;
  unsigned int local_j = threadIdx.x + 1;

  extern __shared__ float sharedmemory[];

  float *sh_h   = sharedmemory;
  float *sh_uh  = sh_h   + (blockDim.y + 2) * (blockDim.x + 2);
  float *sh_vh  = sh_uh  + (blockDim.y + 2) * (blockDim.x + 2);
  float *sh_fh  = sh_vh  + (blockDim.y + 2) * (blockDim.x + 2);
  float *sh_gh  = sh_fh  + (blockDim.y + 2) * (blockDim.x + 2);
  float *sh_fuh = sh_gh  + (blockDim.y + 2) * (blockDim.x + 2);
  float *sh_guh = sh_fuh + (blockDim.y + 2) * (blockDim.x + 2);
  float *sh_fvh = sh_guh + (blockDim.y + 2) * (blockDim.x + 2);
  float *sh_gvh = sh_fvh + (blockDim.y + 2) * (blockDim.x + 2);
  float *sh_hm  = sh_gvh + (blockDim.y + 2) * (blockDim.x + 2);
  float *sh_uhm =  sh_hm + (blockDim.y + 2) * (blockDim.x + 2);
  float *sh_vhm = sh_uhm + (blockDim.y + 2) * (blockDim.x + 2);

  #define SH_ID(local_i, local_j) ((local_i) * (blockDim.x + 2) + (local_j)) 
  #define ID_2D(global_i, global_j) ((global_i) * (nx + 2) + (global_j))

  __syncthreads();

  float programRuntime = 0.0f;

  while (programRuntime < finalRuntime)
  {
    programRuntime += dt;

    writeGlobalToInterior(h, sh_h, global_i, global_j, local_i, local_j, nx, ny, blockDim.x, blockDim.y);
    writeGlobalToInterior(uh, sh_uh, global_i, global_j, local_i, local_j, nx, ny, blockDim.x, blockDim.y);
    writeGlobalToInterior(vh, sh_vh, global_i, global_j, local_i, local_j, nx, ny, blockDim.x, blockDim.y);
    __syncthreads();

    haloExchange(sh_h, sh_uh, sh_vh, h, uh, vh, global_i, global_j, local_i, local_j, nx, ny, blockDim.x, blockDim.y);
    __syncthreads();

    if (global_i < ny + 2 && global_j < nx + 2)
    {
      int local_id = SH_ID(local_i, local_j);

      float g = 9.81f;
      float g_half = 0.5f * g;

      float h_val  = sh_h[local_id];
      float uh_val = sh_uh[local_id];
      float vh_val = sh_vh[local_id];

      float inv_h = 1.0f / h_val;
      float h2 = h_val * h_val;

      sh_fh[local_id] = uh_val;
      sh_gh[local_id] = vh_val;

      float uh2 = uh_val * uh_val;
      float vh2 = vh_val * vh_val;
      float uv = uh_val * vh_val;

      sh_fuh[local_id] = __fmaf_rn(uh2, inv_h, g_half * h2);
      sh_fvh[local_id] = uv * inv_h;

      sh_guh[local_id] = uv * inv_h;
      sh_gvh[local_id] = __fmaf_rn(vh2, inv_h, g_half * h2);
    }
    __syncthreads();

    if (global_i > 0 && global_i < ny + 1 && global_j > 0 && global_j < nx + 1)
    {
      int local_id = SH_ID(local_i, local_j);
      int local_id_left   = SH_ID(local_i, local_j - 1);
      int local_id_right  = SH_ID(local_i, local_j + 1);
      int local_id_bottom = SH_ID(local_i - 1, local_j);
      int local_id_top    = SH_ID(local_i + 1, local_j);

      float h_l  = sh_h[local_id_left];
      float h_r  = sh_h[local_id_right];
      float h_b  = sh_h[local_id_bottom];
      float h_t  = sh_h[local_id_top];

      float uh_l = sh_uh[local_id_left];
      float uh_r = sh_uh[local_id_right];
      float uh_b = sh_uh[local_id_bottom];
      float uh_t = sh_uh[local_id_top];

      float vh_l = sh_vh[local_id_left];
      float vh_r = sh_vh[local_id_right];
      float vh_b = sh_vh[local_id_bottom];
      float vh_t = sh_vh[local_id_top];

      float fh_l = sh_fh[local_id_left];
      float fh_r = sh_fh[local_id_right];
      float gh_b = sh_gh[local_id_bottom];
      float gh_t = sh_gh[local_id_top];

      float fuh_l = sh_fuh[local_id_left];
      float fuh_r = sh_fuh[local_id_right];
      float guh_b = sh_guh[local_id_bottom];
      float guh_t = sh_guh[local_id_top];

      float fvh_l = sh_fvh[local_id_left];
      float fvh_r = sh_fvh[local_id_right];
      float gvh_b = sh_gvh[local_id_bottom];
      float gvh_t = sh_gvh[local_id_top];

      sh_hm[local_id]  = __fmaf_rn(-lambda_x, (fh_r - fh_l),
                       __fmaf_rn(-lambda_y, (gh_t - gh_b),
                       0.25f * (h_l + h_r + h_b + h_t)));

      sh_uhm[local_id] = __fmaf_rn(-lambda_x, (fuh_r - fuh_l),
                       __fmaf_rn(-lambda_y, (guh_t - guh_b),
                       0.25f * (uh_l + uh_r + uh_b + uh_t)));

      sh_vhm[local_id] = __fmaf_rn(-lambda_x, (fvh_r - fvh_l),
                       __fmaf_rn(-lambda_y, (gvh_t - gvh_b),
                       0.25f * (vh_l + vh_r + vh_b + vh_t)));
    }
    __syncthreads();

    if (global_i > 0 && global_i < ny + 1 && global_j > 0 && global_j < nx + 1)
    {
      int local_id = SH_ID(local_i, local_j);

      sh_h[local_id] = sh_hm[local_id];
      sh_uh[local_id] = sh_uhm[local_id];
      sh_vh[local_id] = sh_vhm[local_id];
    }
    __syncthreads();

    applyReflectiveBCs(sh_h, sh_uh, sh_vh, local_i, local_j, blockDim.x, blockDim.y);
    __syncthreads();

    writeInteriorToGlobal(h, sh_h, global_i, global_j, local_i, local_j, nx, ny, blockDim.x, blockDim.y);
    writeInteriorToGlobal(uh, sh_uh, global_i, global_j, local_i, local_j, nx, ny, blockDim.x, blockDim.y);
    writeInteriorToGlobal(vh, sh_vh, global_i, global_j, local_i, local_j, nx, ny, blockDim.x, blockDim.y);
    __syncthreads();
  }

  writeInteriorToGlobal(h, sh_h, global_i, global_j, local_i, local_j, nx, ny, blockDim.x, blockDim.y);
  writeInteriorToGlobal(uh, sh_uh, global_i, global_j, local_i, local_j, nx, ny, blockDim.x, blockDim.y);
  writeInteriorToGlobal(vh, sh_vh, global_i, global_j, local_i, local_j, nx, ny, blockDim.x, blockDim.y);  

  #undef ID_2D
  #undef SH_ID
}
// ****************************************************************************************************************** //

void checkOccupancy() 
{
  int minGridSize = 0;
  int blockSize = 0;
  
  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, shallowWaterSolver, 0, 0);

  std::cout << "Recommended block size: " << blockSize << std::endl;
  std::cout << "Minimum grid size: " << minGridSize << std::endl;
}
// ****************************************************************************************************************** //

// ****************************************************** MAIN ****************************************************** //
int main ( int argc, char *argv[] )
{ 
  // ************************************************** INSTANTIATION ************************************************* //
  int k;

  int nx; 
  int ny; 

  float dx;
  float dy;
  
  float x_length;

  float dt;
  float finalRuntime;
  
  // pointers to host, device memory 
  float *h, *d_h;
  float *uh, *d_uh;
  float *vh, *d_vh;

  float *fh, *d_fh;
  float *fuh, *d_fuh;
  float *fvh, *d_fvh;

  float *gh, *d_gh;
  float *guh, *d_guh;
  float *gvh, *d_gvh;

  float *hm, *d_hm; 
  float *uhm, *d_uhm;
  float *vhm, *d_vhm;

  float *x, *d_x;
  float *y, *d_y;

  // get command line arguments
  getArgs(&nx, &dt, &x_length, &finalRuntime, argc, argv);
  ny = nx; // we assume the grid is square

  // Define the locations of the nodes and time steps and the spacing.
  dx = x_length / ( float ) ( nx );
  dy = x_length / ( float ) ( nx );

  float lambda_x = 0.5f * dt / dx;
  float lambda_y = 0.5f * dt / dy;

  // Define the block and grid sizes
  int dimx = 32;
  int dimy = 20;
  dim3 blockSize(dimx, dimy);
  dim3 gridSize((nx + 2 + blockSize.x - 1) / blockSize.x, (ny + 2 + blockSize.y - 1) / blockSize.y);

  // Calculate shared memory size
  size_t sharedMemSize = ((12 * (blockSize.x+2) * (blockSize.y+2) * sizeof(float)) + 127) & ~127;

  int boundaryBlockSize = 1024;
  int gridSizeY = (ny + boundaryBlockSize - 1) / boundaryBlockSize; 
  int gridSizeX = (nx + boundaryBlockSize - 1) / boundaryBlockSize;  

  // ************************************************ MEMORY ALLOCATIONS ************************************************ //

  // **** Allocate memory on host ****
  // Allocate space (nx+2)((nx+2) long, to account for ghosts
  // height array
  h  = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  hm = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  fh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  gh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );

  // x momentum array
  uh  = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  uhm = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  fuh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  guh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );

  // y momentum array
  vh  = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  vhm = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  fvh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  gvh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );

  // location arrays
  x = ( float * ) malloc ( nx * sizeof ( float ) );
  y = ( float * ) malloc ( ny * sizeof ( float ) );

  // **** Allocate memory on device ****

  // Allocate space (nx+2)((nx+2) long, to account for ghosts
  hipMalloc((void **)&d_h, (nx+2) * (ny+2) * sizeof ( float ));
  hipMalloc((void **)&d_uh, (nx+2) * (ny+2) * sizeof ( float ));
  hipMalloc((void **)&d_vh, (nx+2) * (ny+2) * sizeof ( float ));

  hipMalloc((void **)&d_fh, (nx+2) * (ny+2) * sizeof ( float ));
  hipMalloc((void **)&d_fuh, (nx+2) * (ny+2) * sizeof ( float ));
  hipMalloc((void **)&d_fvh, (nx+2) * (ny+2) * sizeof ( float ));

  hipMalloc((void **)&d_gh, (nx+2) * (ny+2) * sizeof ( float ));
  hipMalloc((void **)&d_guh, (nx+2) * (ny+2) * sizeof ( float ));
  hipMalloc((void **)&d_gvh, (nx+2) * (ny+2) * sizeof ( float ));

  hipMalloc((void **)&d_hm, (nx+2) * (ny+2) * sizeof ( float ));
  hipMalloc((void **)&d_uhm, (nx+2) * (ny+2) * sizeof ( float ));
  hipMalloc((void **)&d_vhm, (nx+2) * (ny+2) * sizeof ( float ));

  hipMalloc((void **)&d_x, nx * sizeof ( float ));
  hipMalloc((void **)&d_y, ny * sizeof ( float ));

  hipMemset(d_h, 0, (nx+2) * (ny+2) * sizeof ( float ));
  hipMemset(d_uh, 0, (nx+2) * (ny+2) * sizeof ( float ));
  hipMemset(d_vh, 0, (nx+2) * (ny+2) * sizeof ( float ));
  
  // *********************************************************************** INITIAL CONDITIONS ********************************************************************** //

  printf ( "\n" );
  printf ( "SHALLOW_WATER_2D\n" );
  printf ( "\n" );

  checkOccupancy();
  
  for(k = 1; k < 6; k++)
  {
    // Apply the initial conditions.
    initializeInterior<<<gridSize, blockSize>>>(d_x, d_y, d_h, d_uh, d_vh, nx, ny, dx, dy, x_length);

    applyLeftBoundary<<<gridSizeY, boundaryBlockSize>>>(d_h, d_uh, d_vh, nx, ny);

    applyRightBoundary<<<gridSizeY, boundaryBlockSize>>>(d_h, d_uh, d_vh, nx, ny);

    applyBottomBoundary<<<gridSizeX, boundaryBlockSize>>>(d_h, d_uh, d_vh, nx, ny);

    applyTopBoundary<<<gridSizeX, boundaryBlockSize>>>(d_h, d_uh, d_vh, nx, ny);

    hipDeviceSynchronize();

    if(k == 1 && nx == 200)
    {
      hipMemcpy(h, d_h, (nx+2) * (ny+2) * sizeof ( float ), hipMemcpyDeviceToHost);
      hipMemcpy(uh, d_uh, (nx+2) * (ny+2) * sizeof ( float ), hipMemcpyDeviceToHost);
      hipMemcpy(vh, d_vh, (nx+2) * (ny+2) * sizeof ( float ), hipMemcpyDeviceToHost);

      hipMemcpy(x, d_x, nx * sizeof ( float ), hipMemcpyDeviceToHost);
      hipMemcpy(y, d_y, ny * sizeof ( float ), hipMemcpyDeviceToHost);

      // Write initial condition to a file
      writeResults(h, uh, vh, x, y, 0.000000, nx, ny);
    }

    // ******************************************************************** COMPUTATION SECTION ******************************************************************** //

    // start program timer
    auto start_time = std::chrono::steady_clock::now();

    shallowWaterSolver<<<gridSize, blockSize, sharedMemSize>>>(d_h, d_uh, d_vh, lambda_x, lambda_y, nx, ny, dt, finalRuntime);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
    {
      printf("CUDA Error launching shallowWaterSolver: %s\n", hipGetErrorString(err));
    }

    hipDeviceSynchronize();  // Wait for kernel to finish

    // stop timer
    auto end_time = std::chrono::steady_clock::now();
    std::chrono::duration<double> time_elapsed = end_time - start_time;

    // Print out the results
    printf("Problem size: %d, Iteration: %d, Elapsed time: %f s\n", nx, k, time_elapsed);

    if(k == 1 && nx == 200)
    {
      hipMemcpy(h, d_h, (nx+2) * (ny+2) * sizeof ( float ), hipMemcpyDeviceToHost);
      hipMemcpy(uh, d_uh, (nx+2) * (ny+2) * sizeof ( float ), hipMemcpyDeviceToHost);
      hipMemcpy(vh, d_vh, (nx+2) * (ny+2) * sizeof ( float ), hipMemcpyDeviceToHost);

      // Write initial condition to a file
      writeResults(h, uh, vh, x, y, 0.500000, nx, ny);
    }
  }

  // ******************************************************************** DEALLOCATE MEMORY ******************************************************************** //

  //Free device memory.
  hipFree(d_h);
  hipFree(d_uh);
  hipFree(d_vh);

  hipFree(d_fh);
  hipFree(d_fuh);
  hipFree(d_fvh);

  hipFree(d_gh);
  hipFree(d_guh);
  hipFree(d_gvh);

  hipFree(d_hm);
  hipFree(d_uhm);
  hipFree(d_vhm);

  hipFree(d_x);
  hipFree(d_y);

  // Free host memory.
  free ( h );
  free ( uh );
  free ( vh ); 

  free ( fh );
  free ( fuh );
  free ( fvh );

  free ( gh );
  free ( guh );
  free ( gvh ); 

  free ( x );
  free ( y );

  // Terminate.
  printf ( "\n" );
  printf ( "SHALLOW_WATER_2D:\n" );
  printf ( "Normal end of execution.\n" );
  printf ( "\n" );

  return 0;
}
// ******************************************************************************************************************************************** //