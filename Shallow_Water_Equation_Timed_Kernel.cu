#include "hip/hip_runtime.h"
# include "common.h"
# include <stdlib.h>
# include <stdio.h>
# include <math.h>
# include <string.h>
# include <time.h>
#include <chrono>
# include <hip/hip_runtime.h>

# define ID_2D(i,j,nx) ((i)*(nx+2)+(j))

//************************************************ UTILITIES ************************************************//

void getArgs(int *nx, double *dt, float *x_length, double *t_final, int argc, char *argv[])
{
  // Get the quadrature file root name:

  if ( argc <= 1 ){
    *nx = 400;
  }else{
    *nx = atoi ( argv[1] );
  }
  
  if ( argc <= 2 ){
    *dt = 0.002;
  }else{
    *dt = atof ( argv[2] );
  }
  
  if ( argc <= 3 ){
    *x_length = 10.0;
  }else{
    *x_length = atof ( argv[3] );
    }
  
  if ( argc <= 4 ){
    *t_final = 0.5;
  }else{
    *t_final = atof ( argv[4] );
  }
}
/******************************************************************************/

void writeResults(float h[], float uh[], float vh[], float x[], float y[], float time, int nx, int ny)
{
  char filename[50];

  int i, j, id;

  //Create the filename based on the time step.
  sprintf(filename, "tc2d_%08.6f.dat", time);

  //Open the file.
  FILE *file = fopen (filename, "wt" );
    
  if (!file)
  {
    fprintf (stderr, "\n" );

    fprintf (stderr, "WRITE_RESULTS - Fatal error!\n");

    fprintf (stderr, "  Could not open the output file.\n");

    exit (1);
  }

  else
  {  
    //Write the data.
    for ( i = 0; i < ny; i++ ) 
      for ( j = 0; j < nx; j++ )
      {
        id = ID_2D(i + 1, j + 1, nx);
        fprintf ( file, "%24.16g\t%24.16g\t%24.16g\t %24.16g\t %24.16g\n", x[j], y[i], h[id], uh[id], vh[id]);
      }
    
    //Close the file.
    fclose (file);
  }

  return;
}
/******************************************************************************/

void initial_conditions(int nx, int ny, float dx, float dy,  float x_length, float x[],float y[], float h[], float uh[] ,float vh[])
{
  int i,j, id, id1;

  for ( i = 1; i < nx+1; i++ )
    {
      x[i-1] = -x_length/2+dx/2+(i-1)*dx;
      y[i-1] = -x_length/2+dy/2+(i-1)*dy;
    }

  for ( i = 1; i < nx+1; i++ )
    for( j = 1; j < ny+1; j++)
    {
      float xx = x[j-1];
      float yy = y[i-1];
      id=ID_2D(i,j,nx);
      h[id] = 1.0 + 0.4*exp ( -5 * ( xx*xx + yy*yy) );
    }
  
  for ( i = 1; i < nx+1; i++ )
    for( j = 1; j < ny+1; j++)
    {
      id=ID_2D(i,j,nx);
      uh[id] = 0.0;
      vh[id] = 0.0;
    }

  //set boundaries
  //bottom
  i=0;
  for( j = 1; j < nx+1; j++)
    {
      id=ID_2D(i,j,nx);
      id1=ID_2D(i+1,j,nx);

      h[id] = h[id1];
      uh[id] = 0.0;
      vh[id] = 0.0;
    }

  //top
  i=nx+1;
  for( j = 1; j < nx+1; j++)
    {
      id=ID_2D(i,j,nx);
      id1=ID_2D(i-1,j,nx);

      h[id] = h[id1];
      uh[id] = 0.0;
      vh[id] = 0.0;
    }

  //left
  j=0;
  for( i = 1; i < ny+1; i++)
    {
      id=ID_2D(i,j,nx);
      id1=ID_2D(i,j+1,nx);

      h[id] = h[id1];
      uh[id] = 0.0;
      vh[id] = 0.0;
    }

  //right
  j=nx+1;
  for( i = 1; i < ny+1; i++)
    {
      id=ID_2D(i,j,nx);
      id1=ID_2D(i,j-1,nx);

      h[id] = h[id1];
      uh[id] = 0.0;
      vh[id] = 0.0;
    }

  return;
}
/******************************************************************************/

__global__ void computeFluxesGPU(float *h, float *uh, float *vh, float *fh, float *fuh, float *fvh, float *gh, float *guh, float *gvh, int nx, int ny) 
{
  unsigned int i = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned int j = threadIdx.x + blockIdx.x * blockDim.x;
  
  unsigned int id = ((i) * (nx + 2) + (j));

  float g = 9.81f; // Gravitational acceleration
  float h_safe = fmaxf(h[id], 1e-6f); // Prevent division by zero
  
  if (i < ny + 2 && j < nx + 2)
  {
    // Compute fluxes safely
    fh[id] = uh[id];

    fuh[id] = uh[id] * uh[id] / h_safe + 0.5f * g * h_safe * h_safe;

    fvh[id] = uh[id] * vh[id] / h_safe;

    gh[id] = vh[id];

    guh[id] = uh[id] * vh[id] / h_safe;

    gvh[id] = vh[id] * vh[id] / h_safe + 0.5f * g * h_safe * h_safe;
  }
}
/******************************************************************************/

__global__ void computeVariablesGPU(float *hm, float *uhm, float *vhm, float *fh, float *fuh, float *fvh, float *gh, float *guh, float *gvh, float *h, float *uh, float *vh, float lambda_x, float lambda_y, int nx, int ny)
{
  unsigned int i = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned int j = threadIdx.x + blockIdx.x * blockDim.x;
  
  unsigned int id = ((i) * (nx + 2) + (j));

  unsigned int id_left   = ((i) * (nx + 2) + (j - 1));
  unsigned int id_right  = ((i) * (nx + 2) + (j + 1));
  unsigned int id_bottom = ((i - 1) * (nx + 2) + (j));
  unsigned int id_top    = ((i + 1) * (nx + 2) + (j));

  if (i > 0 && i < ny + 1 && j > 0 && j < nx + 1)  // Ensure proper bounds
  {
    hm[id] = 0.25 * (h[id_left] + h[id_right] + h[id_bottom] + h[id_top])
          - lambda_x * (fh[id_right] - fh[id_left])
          - lambda_y * (gh[id_top] - gh[id_bottom]);

    uhm[id] = 0.25 * (uh[id_left] + uh[id_right] + uh[id_bottom] + uh[id_top])
            - lambda_x * (fuh[id_right] - fuh[id_left])
            - lambda_y * (guh[id_top] - guh[id_bottom]);

    vhm[id] = 0.25 * (vh[id_left] + vh[id_right] + vh[id_bottom] + vh[id_top])
            - lambda_x * (fvh[id_right] - fvh[id_left])
            - lambda_y * (gvh[id_top] - gvh[id_bottom]);
  }
}
/******************************************************************************/

__global__ void updateVariablesGPU(float *h, float *uh, float *vh, float *hm, float *uhm, float *vhm, int nx, int ny)
{
  unsigned int i = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned int j = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int id = ((i) * (nx + 2) + (j));

  if (i > 0 && i < ny + 1 && j > 0 && j < nx + 1)  // Ensure proper bounds
  {
    h[id] = hm[id];
    uh[id] = uhm[id];
    vh[id] = vhm[id];
  }
}
/******************************************************************************/

__global__ void applyBoundaryConditionsGPU(float *h, float *uh, float *vh, int nx, int ny)
{
  unsigned int i = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned int j = threadIdx.x + blockIdx.x * blockDim.x;

  unsigned int id_ghost;

  unsigned int id = ((i) * (nx + 2) + (j));
  unsigned int id_left   = ((i) * (nx + 2) + (j - 1));
  unsigned int id_right  = ((i) * (nx + 2) + (j + 1));
  unsigned int id_bottom = ((i - 1) * (nx + 2) + (j));
  unsigned int id_top    = ((i + 1) * (nx + 2) + (j));

  // Left Boundary (j = 1) - Reflective
  if (j == 1 && i > 0 && i < ny + 1) 
  {
    h[id_left]  = h[id];
    uh[id_left] = -uh[id];  // Flip x-momentum
    vh[id_left] = vh[id];   
  }

  // Right Boundary (j = nx) - Reflective
  if (j == nx && i > 0 && i < ny + 1) 
  {
    h[id_right]  = h[id];
    uh[id_right] = -uh[id];  // Flip x-momentum
    vh[id_right] = vh[id];   
  }

  // Bottom Boundary (i = 1) - Reflective
  if (i == 1 && j > 0 && j < nx + 1) 
  {
    h[id_bottom]  = h[id];
    uh[id_bottom] = uh[id];   
    vh[id_bottom] = -vh[id];  // Flip y-momentum
  }

  // Top Boundary (i = ny) - Reflective
  if (i == ny && j > 0 && j < nx + 1) 
  {
    h[id_top]  = h[id];
    uh[id_top] = uh[id];   
    vh[id_top] = -vh[id];  // Flip y-momentum
  }
}
// ****************************************************************************************************************** //

// ****************************************************** MAIN ****************************************************** //
int main ( int argc, char *argv[] )
{ 
// ************************************************** INSTANTIATION ************************************************* //
  int k, l;
  int nx; 
  int ny; 

  float *x;
  float *y;

  float dx;
  float dy;
  float x_length;

  double dt;
  double time; 
  double t_final;
  
  // pointers to host, device memory 
  float *h, *d_h;
  float *uh, *d_uh;
  float *vh, *d_vh;

  float *fh, *d_fh;
  float *fuh, *d_fuh;
  float *fvh, *d_fvh;

  float *gh, *d_gh;
  float *guh, *d_guh;
  float *gvh, *d_gvh;

  float *hm, *d_hm; 
  float *uhm, *d_uhm;
  float *vhm, *d_vhm;

  // get command line arguments
  getArgs(&nx, &dt, &x_length, &t_final, argc, argv);
  ny = nx; // we assume this, does not have to be this way

  // Define the locations of the nodes and time steps and the spacing.
  dx = x_length / ( float ) ( nx );
  dy = x_length / ( float ) ( nx );

  float lambda_x = 0.5  * (float) dt / dx;
  float lambda_y = 0.5 * (float) dt / dy;

  // Define the block and grid sizes
  int dimx = 32;
  int dimy = 32;
  dim3 blockSize(dimx, dimy);
  dim3 gridSize((nx + 2 + blockSize.x - 1) / blockSize.x, (ny + 2 + blockSize.y - 1) / blockSize.y);

  // ************************************************ MEMORY ALLOCATIONS ************************************************ //

  // **** Allocate memory on host ****
  // Allocate space (nx+2)((nx+2) long, to account for ghosts
  // height array
  h  = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  hm = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  fh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  gh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );

  // x momentum array
  uh  = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  uhm = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  fuh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  guh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );

  // y momentum array
  vh  = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  vhm = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  fvh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  gvh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );

  // location arrays
  x = ( float * ) malloc ( nx * sizeof ( float ) );
  y = ( float * ) malloc ( ny * sizeof ( float ) );

  // **** Allocate memory on device ****

  // Allocate space (nx+2)((nx+2) long, to account for ghosts
  CHECK(hipMalloc((void **)&d_h, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_uh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_vh, (nx+2)*(ny+2) * sizeof ( float )));

  CHECK(hipMalloc((void **)&d_fh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_fuh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_fvh, (nx+2)*(ny+2) * sizeof ( float )));

  CHECK(hipMalloc((void **)&d_gh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_guh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_gvh, (nx+2)*(ny+2) * sizeof ( float )));

  CHECK(hipMalloc((void **)&d_hm, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_uhm, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_vhm, (nx+2)*(ny+2) * sizeof ( float )));

  // *********************************************************************** INITIAL CONDITIONS ********************************************************************** //

  printf ( "\n" );
  printf ( "SHALLOW_WATER_2D\n" );
  printf ( "\n" );
  
  for(k = 1; k < 11; k++)
  {
    // set time to zero and step counter to zero
    time = 0.0f;
    l = 0;
    
    // instantiate section timing variables
    double time_elapsed_cf = 0.0;
    double time_elapsed_cv = 0.0;
    double time_elapsed_uv = 0.0;
    double time_elapsed_bc = 0.0;

    initial_conditions(nx, ny, dx, dy, x_length, x, y, h, uh, vh);

    // Move data to the device for calculations
    CHECK(hipMemcpy(d_h, h, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_uh, uh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_vh, vh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyHostToDevice));

    // ******************************************************************** COMPUTATION SECTION ******************************************************************** //

    // start timer
    auto start_time = std::chrono::steady_clock::now();

    while (time < t_final) // time loop begins
    {
      // Take a time step and increase step counter
      time = time + dt;
      l++;

      // *********************************************************************************************************************************************************** //

      // Start timing compute fluxes calculations
      auto start_time_cf = std::chrono::steady_clock::now();

      // **** COMPUTE FLUXES ****
      computeFluxesGPU<<<gridSize, blockSize>>>(d_h, d_uh, d_vh, d_fh, d_fuh, d_fvh, d_gh, d_guh, d_gvh, nx, ny);

      // Stop timing compute fluxes calculations
      auto end_time_cf = std::chrono::steady_clock::now();

      // calculate time elapsed for compute fluxes
      time_elapsed_cf = time_elapsed_cf + std::chrono::duration<double>(end_time_cf - start_time_cf).count();

      // *********************************************************************************************************************************************************** //

      // Start timing compute variable calculations
      auto start_time_cv = std::chrono::steady_clock::now();
      
      // **** COMPUTE VARIABLES ****
      computeVariablesGPU<<<gridSize, blockSize>>>(d_hm, d_uhm, d_vhm, d_fh, d_fuh, d_fvh, d_gh, d_guh, d_gvh, d_h, d_uh, d_vh, lambda_x, lambda_y, nx, ny);
    
      // Stop timing compute variable calculations
      auto end_time_cv = std::chrono::steady_clock::now();

      // calculate time elapsed for compute variables
      time_elapsed_cv = time_elapsed_cv + std::chrono::duration<double>(end_time_cv - start_time_cv).count();
      // *********************************************************************************************************************************************************** //

      // Start timing update variables calculations
      auto start_time_uv = std::chrono::steady_clock::now();

      // **** UPDATE VARIABLES ****
      updateVariablesGPU<<<gridSize, blockSize>>>(d_h, d_uh, d_vh, d_hm, d_uhm, d_vhm, nx, ny);

      // Stop timing update variables calculations
      auto end_time_uv = std::chrono::steady_clock::now();

      // calculate time elapsed for update variables
      time_elapsed_uv = time_elapsed_uv + std::chrono::duration<double>(end_time_uv - start_time_uv).count();

      // *********************************************************************************************************************************************************** //

      // Start timing apply boundary condition calculations
      auto start_time_bc = std::chrono::steady_clock::now();

      // **** APPLY BOUNDARY CONDITIONS ****
      applyBoundaryConditionsGPU<<<gridSize, blockSize>>>(d_h, d_uh, d_vh, nx, ny);  

      // Stop timing apply boundary condition calculations
      auto end_time_bc = std::chrono::steady_clock::now();

      // calculate time elapsed for apply boundary conditions
      time_elapsed_bc = time_elapsed_bc + std::chrono::duration<double>(end_time_bc - start_time_bc).count();

    } // end time loop

    // stop timer
    auto end_time = std::chrono::steady_clock::now();
    std::chrono::duration<double> time_elapsed = end_time - start_time;
    
    double avg_time_elapsed_cf = time_elapsed_cf / (double) l;
    double avg_time_elapsed_cv = time_elapsed_cv / (double) l;
    double avg_time_elapsed_uv = time_elapsed_uv / (double) l;
    double avg_time_elapsed_bc = time_elapsed_bc / (double) l;

    // Print out the results
    printf("Problem size: %d, Time steps: %d, Iteration: %d, Elapsed time: %f s, Average elapsed time for compute fluxes: %f s, Average elapsed time for compute variables: %f s, Average elapsed time for update variables: %f s, Average elapsed time for apply boundary conditions: %f s\n", nx, l, k, time_elapsed, avg_time_elapsed_cf, avg_time_elapsed_cv, avg_time_elapsed_uv, avg_time_elapsed_bc);
  }

  // ******************************************************************** DEALLOCATE MEMORY ******************************************************************** //

  //Free device memory.
  CHECK(hipFree(d_h));
  CHECK(hipFree(d_uh));
  CHECK(hipFree(d_vh));

  CHECK(hipFree(d_fh));
  CHECK(hipFree(d_fuh));
  CHECK(hipFree(d_fvh));

  CHECK(hipFree(d_gh));
  CHECK(hipFree(d_guh));
  CHECK(hipFree(d_gvh));

  CHECK(hipFree(d_hm));
  CHECK(hipFree(d_uhm));
  CHECK(hipFree(d_vhm));

  // Free host memory.
  free ( h );
  free ( uh );
  free ( vh ); 

  free ( fh );
  free ( fuh );
  free ( fvh );

  free ( gh );
  free ( guh );
  free ( gvh ); 

  free ( x );
  free ( y );

  // Terminate.
  printf ( "\n" );
  printf ( "SHALLOW_WATER_2D:\n" );
  printf ( "Normal end of execution.\n" );
  printf ( "\n" );

  return 0;
}
// ******************************************************************************************************************************************** //