#include "hip/hip_runtime.h"
# include "common.h"
# include <stdlib.h>
# include <stdio.h>
# include <math.h>
# include <string.h>
# include <time.h>
#include <chrono>
# include <hip/hip_runtime.h>

#define ID_2D(i,j,nx) ((i)*(nx+2)+(j))

//************************************************ UTILITIES ************************************************//

void getArgs(int *nx, double *dt, float *x_length, double *t_final, int argc, char *argv[])
{
  // Get the quadrature file root name:

  if ( argc <= 1 ){
    *nx = 400;
  }else{
    *nx = atoi ( argv[1] );
  }
  
  if ( argc <= 2 ){
    *dt = 0.002;
  }else{
    *dt = atof ( argv[2] );
  }
  
  if ( argc <= 3 ){
    *x_length = 10.0;
  }else{
    *x_length = atof ( argv[3] );
    }
  
  if ( argc <= 4 ){
    *t_final = 0.5;
  }else{
    *t_final = atof ( argv[4] );
  }
}
// ****************************************************************************** //

void writeResults(float h[], float uh[], float vh[], float x[], float y[], float time, int nx, int ny)
{
  char filename[50];

  int i, j, id;

  //Create the filename based on the time step.
  sprintf(filename, "tc2d_%08.6f.dat", time);

  //Open the file.
  FILE *file = fopen (filename, "wt" );
    
  if (!file)
  {
    fprintf (stderr, "\n" );

    fprintf (stderr, "WRITE_RESULTS - Fatal error!\n");

    fprintf (stderr, "  Could not open the output file.\n");

    exit (1);
  }

  else
  {  
    //Write the data.
    for ( i = 0; i < ny; i++ ) 
      for ( j = 0; j < nx; j++ )
      {
        id = ID_2D(i + 1, j + 1, nx);
        fprintf ( file, "%24.16g\t%24.16g\t%24.16g\t %24.16g\t %24.16g\n", x[j], y[i], h[id], uh[id], vh[id]);
      }
    
    //Close the file.
    fclose (file);
  }

  return;
}
// ****************************************************************************** //

void initialConditions(int nx, int ny, float dx, float dy,  float x_length, float x[], float y[], float h[], float uh[] ,float vh[])
{
  int i,j, id, id1;

  for ( i = 1; i < ny + 2; i++ )
    for (j = 1; j < nx + 2; j++)
    {
      x[j-1] = -x_length/2+dx/2+(j-1)*dx;
      y[i-1] = -x_length/2+dy/2+(i-1)*dy;
    }

  for ( i = 1; i < ny+1; i++ )
    for( j = 1; j < nx+1; j++)
    {
      float xx = x[j-1];
      float yy = y[i-1];
      id=ID_2D(i,j,nx);
      h[id] = 1.0 + 0.4 * exp ( -15 * ( xx*xx + yy*yy) );
    }
  
  for ( i = 1; i < ny+1; i++ )
    for( j = 1; j < nx+1; j++)
    {
      id=ID_2D(i,j,nx);
      uh[id] = 0.0;
      vh[id] = 0.0;
    }

  //set boundaries
  //bottom
  i=0;
  for( j = 1; j < nx+1; j++)
    {
      id=ID_2D(i,j,nx);
      id1=ID_2D(i+1,j,nx);

      h[id] = h[id1];
      uh[id] = 0.0;
      vh[id] = 0.0;
    }

  //top
  i=nx+1;
  for( j = 1; j < nx+1; j++)
    {
      id=ID_2D(i,j,nx);
      id1=ID_2D(i-1,j,nx);

      h[id] = h[id1];
      uh[id] = 0.0;
      vh[id] = 0.0;
    }

  //left
  j=0;
  for( i = 1; i < ny+1; i++)
    {
      id=ID_2D(i,j,nx);
      id1=ID_2D(i,j+1,nx);

      h[id] = h[id1];
      uh[id] = 0.0;
      vh[id] = 0.0;
    }

  //right
  j=nx+1;
  for( i = 1; i < ny+1; i++)
    {
      id=ID_2D(i,j,nx);
      id1=ID_2D(i,j-1,nx);

      h[id] = h[id1];
      uh[id] = 0.0;
      vh[id] = 0.0;
    }

  return;
}
// ****************************************************************************** //

void generateDrops( int nx, int ny, float x[], float y[], float h[], float uh[], float vh[])
{
  int i, j, id;

  unsigned int randNumber;

  // Determine a section's grid size
  // This value will be used to section off the nx x nx grid into 16 sections
  unsigned int sectionSquareLength = (nx * ny) / 16;

  // Generate a random number between 0 & 15
  randNumber = rand() % 16;

  // Determine section bounds based on random number
  unsigned int sectionStart = randNumber * sectionSquareLength;
  unsigned int sectionEnd = (randNumber + 1) * sectionSquareLength;

  for (i = sectionStart + 1; i < sectionEnd + 1; i++)
    for (j = sectionStart + 1; j < sectionEnd + 1; j++)
    {
      id = ID_2D(i,j,nx);
      
      float xx = x[j - 1];
      float yy = y[i - 1];

      h[id] += 0.4f * expf(-15 * ( xx*xx + yy*yy));

      // Sample momentum from neighboring points 
      int id_left  = ID_2D(i, j - 1, nx);
      int id_right = ID_2D(i, j + 1, nx);
      int id_up    = ID_2D(i - 1, j, nx);
      int id_down  = ID_2D(i + 1, j, nx);

      uh[id] = (uh[id_left] + uh[id_right] + uh[id_up] + uh[id_down]) / 4.0f;
      vh[id] = (vh[id_left] + vh[id_right] + vh[id_up] + vh[id_down]) / 4.0f;
    }
}
// ****************************************************************************** //

__global__ void computeFluxesGPU(float *h, float *uh, float *vh, float *fh, float *fuh, float *fvh, float *gh, float *guh, float *gvh, int nx, int ny) 
{
  unsigned int i = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned int j = threadIdx.x + blockIdx.x * blockDim.x;
  
  unsigned int id = ((i) * (nx + 2) + (j));

  float g = 9.81f; // Gravitational acceleration
  float h_safe = fmaxf(h[id], 1e-6f); // Prevent division by zero
  
  if (i < ny + 2 && j < nx + 2)
  {
    // Compute fluxes safely
    fh[id] = uh[id];

    fuh[id] = uh[id] * uh[id] / h_safe + 0.5f * g * h_safe * h_safe;

    fvh[id] = uh[id] * vh[id] / h_safe;

    gh[id] = vh[id];

    guh[id] = uh[id] * vh[id] / h_safe;

    gvh[id] = vh[id] * vh[id] / h_safe + 0.5f * g * h_safe * h_safe;
  }
}
// ****************************************************************************** //

__global__ void computeVariablesGPU(float *hm, float *uhm, float *vhm, float *fh, float *fuh, float *fvh, float *gh, float *guh, float *gvh, float *h, float *uh, float *vh, float lambda_x, float lambda_y, int nx, int ny)
{
  unsigned int i = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned int j = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int id, id_left, id_right, id_bottom, id_top;

  if (i > 0 && i < ny + 1 && j > 0 && j < nx + 1)  // Ensure proper bounds
  {
    id = ((i) * (nx + 2) + (j));

    id_left   = ((i) * (nx + 2) + (j - 1));
    id_right  = ((i) * (nx + 2) + (j + 1));
    id_bottom = ((i - 1) * (nx + 2) + (j));
    id_top    = ((i + 1) * (nx + 2) + (j));

    hm[id] = 0.25 * (h[id_left] + h[id_right] + h[id_bottom] + h[id_top])
          - lambda_x * (fh[id_right] - fh[id_left])
          - lambda_y * (gh[id_top] - gh[id_bottom]);

    uhm[id] = 0.25 * (uh[id_left] + uh[id_right] + uh[id_bottom] + uh[id_top])
            - lambda_x * (fuh[id_right] - fuh[id_left])
            - lambda_y * (guh[id_top] - guh[id_bottom]);

    vhm[id] = 0.25 * (vh[id_left] + vh[id_right] + vh[id_bottom] + vh[id_top])
            - lambda_x * (fvh[id_right] - fvh[id_left])
            - lambda_y * (gvh[id_top] - gvh[id_bottom]);
  }
}
// ****************************************************************************** //

__global__ void updateVariablesGPU(float *h, float *uh, float *vh, float *hm, float *uhm, float *vhm, int nx, int ny)
{
  unsigned int i = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned int j = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int id;

  if (i > 0 && i < ny + 1 && j > 0 && j < nx + 1)  // Ensure proper bounds
  {
    id = ((i) * (nx + 2) + (j));

    h[id] = hm[id];
    uh[id] = uhm[id];
    vh[id] = vhm[id];
  }
}
// ****************************************************************************** //

__global__ void applyBoundaryConditionsGPU(float *h, float *uh, float *vh, int nx, int ny, int bc_type)
{
  unsigned int i = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned int j = threadIdx.x + blockIdx.x * blockDim.x;

  unsigned int id, id_ghost;

  if (bc_type == 1) // Dirichlet Boundary Conditions
  {  
    // Left Boundary (j = 0)
    if (j == 0 && i >= 1 && i <= ny) 
    {
      id = ID_2D(i, 1, nx);
      id_ghost = ID_2D(i, 0, nx);
      h[id_ghost]  = h[id];
      uh[id_ghost] = uh[id];
      vh[id_ghost] = vh[id];
    }

    // Right Boundary (j = nx + 1)
    if (j == nx + 1 && i >= 1 && i <= ny) 
    {
      id = ID_2D(i, nx, nx);
      id_ghost = ID_2D(i, nx + 1, nx);
      h[id_ghost]  = h[id];
      uh[id_ghost] = uh[id];
      vh[id_ghost] = vh[id];
    }

    // Bottom Boundary (i = 0)
    if (i == 0 && j >= 1 && j <= nx) 
    {
      id = ID_2D(1, j, nx);
      id_ghost = ID_2D(0, j, nx);
      h[id_ghost]  = h[id];
      uh[id_ghost] = uh[id];
      vh[id_ghost] = vh[id];
    }

    // Top Boundary (i = ny + 1)
    if (i == ny + 1 && j >= 1 && j <= nx) 
    {
      id = ID_2D(ny, j, nx);
      id_ghost = ID_2D(ny + 1, j, nx);
      h[id_ghost]  = h[id];
      uh[id_ghost] = uh[id];
      vh[id_ghost] = vh[id];
    }
  }

  else if (bc_type == 2) // Periodic Boundary Conditions
  {  
    // Left to Right Periodic Boundary (wraps leftmost to rightmost)
    if (j == 0 && i >= 1 && i <= ny) 
    {
      id = ID_2D(i, nx, nx);
      id_ghost = ID_2D(i, 0, nx);
      h[id_ghost]  = h[id];
      uh[id_ghost] = uh[id];
      vh[id_ghost] = vh[id];
    }

    // Right to Left Periodic Boundary (wraps rightmost to leftmost)
    if (j == nx + 1 && i >= 1 && i <= ny) 
    {
      id = ID_2D(i, 1, nx);
      id_ghost = ID_2D(i, nx + 1, nx);
      h[id_ghost]  = h[id];
      uh[id_ghost] = uh[id];
      vh[id_ghost] = vh[id];
    }

    // Bottom to Top Periodic Boundary (wraps bottom to top)
    if (i == 0 && j >= 1 && j <= nx) 
    {
      id = ID_2D(ny, j, nx);
      id_ghost = ID_2D(0, j, nx);
      h[id_ghost]  = h[id];
      uh[id_ghost] = uh[id];
      vh[id_ghost] = vh[id];
    }

    // Top to Bottom Periodic Boundary (wraps top to bottom)
    if (i == ny + 1 && j >= 1 && j <= nx) 
    {
      id = ID_2D(1, j, nx);
      id_ghost = ID_2D(ny + 1, j, nx);
      h[id_ghost]  = h[id];
      uh[id_ghost] = uh[id];
      vh[id_ghost] = vh[id];
    }  
  }
  else if (bc_type == 3) // Reflective Boundary Conditions
  {  
    // Left Boundary (j = 1) - Reflective
    if (j == 1 && i > 0 && i < ny + 1) 
    {
      id = ((i) * (nx + 2) + (j));
      id_ghost = ((i) * (nx + 2) + (j - 1));
      h[id_ghost]  = h[id];
      uh[id_ghost] = -uh[id];  // Flip normal velocity
      vh[id_ghost] = vh[id];   // Keep tangential velocity
    }

    // Right Boundary (j = nx) - Reflective
    if (j == nx && i > 0 && i < ny + 1) 
    {
      id = ((i) * (nx + 2) + (j));
      id_ghost = ((i) * (nx + 2) + (j + 1));
      h[id_ghost]  = h[id];
      uh[id_ghost] = -uh[id];  // Flip normal velocity
      vh[id_ghost] = vh[id];   // Keep tangential velocity
    }

    // Bottom Boundary (i = 1) - Reflective
    if (i == 1 && j > 0 && j < nx + 1) 
    {
      id = ((i) * (nx + 2) + (j));
      id_ghost = ((i - 1) * (nx + 2) + (j));
      h[id_ghost]  = h[id];
      uh[id_ghost] = uh[id];   // Keep tangential velocity
      vh[id_ghost] = -vh[id];  // Flip normal velocity
    }

    // Top Boundary (i = ny) - Reflective
    if (i == ny && j > 0 && j < nx + 1) 
    {
      id = ((i) * (nx + 2) + (j));
      id_ghost = ((i + 1) * (nx + 2) + (j));
      h[id_ghost]  = h[id];
      uh[id_ghost] = uh[id];   // Keep tangential velocity
      vh[id_ghost] = -vh[id];  // Flip normal velocity
    }
  }
}
// ****************************************************************************************************************** //

// ****************************************************** MAIN ****************************************************** //
int main ( int argc, char *argv[] )
{ 
// ************************************************** INSTANTIATION ************************************************* //
  
  unsigned int randNumber;
  unsigned int timeSeed;

  int k;
  int nx; 
  int ny; 

  float *x, *d_x;
  float *y, *d_y;

  float dx;
  float dy;
  float x_length;

  double dt;
  double programRuntime; 
  double finalRuntime;

  // pointers to host, device memory 
  float *h, *d_h;
  float *uh, *d_uh;
  float *vh, *d_vh;

  float *fh, *d_fh;
  float *fuh, *d_fuh;
  float *fvh, *d_fvh;

  float *gh, *d_gh;
  float *guh, *d_guh;
  float *gvh, *d_gvh;

  float *hm, *d_hm; 
  float *uhm, *d_uhm;
  float *vhm, *d_vhm;

  // get command line arguments
  getArgs(&nx, &dt, &x_length, &finalRuntime, argc, argv);
  ny = nx; // we assume this, does not have to be this way

  // Define the locations of the nodes and time steps and the spacing.
  dx = x_length / ( float ) ( nx );
  dy = x_length / ( float ) ( nx );

  float lambda_x = 0.5  * (float) dt / dx;
  float lambda_y = 0.5 * (float) dt / dy;

  // Define the block and grid sizes
  int dimx = 32;
  int dimy = 32;
  dim3 blockSize(dimx, dimy);
  dim3 gridSize((nx + 2 + blockSize.x - 1) / blockSize.x, (ny + 2 + blockSize.y - 1) / blockSize.y);

  timeSeed = time(NULL);
  srand(timeSeed);

  // ************************************************ MEMORY ALLOCATIONS ************************************************ //

  // **** Allocate memory on host ****
  // Allocate space (nx+2)((nx+2) long, to account for ghosts
  // height array
  h  = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  hm = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  fh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  gh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );

  // x momentum array
  uh  = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  uhm = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  fuh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  guh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );

  // y momentum array
  vh  = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  vhm = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  fvh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );
  gvh = ( float * ) malloc ( (nx+2)*(ny+2) * sizeof ( float ) );

  // location arrays
  x = ( float * ) malloc ( nx * sizeof ( float ) );
  y = ( float * ) malloc ( ny * sizeof ( float ) );

  // **** Allocate memory on device ****

  CHECK(hipMalloc((void **)&d_x, nx * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_y, ny * sizeof ( float )));

  // Allocate space (nx+2)((nx+2) long, to account for ghosts
  CHECK(hipMalloc((void **)&d_h, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_uh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_vh, (nx+2)*(ny+2) * sizeof ( float )));

  CHECK(hipMalloc((void **)&d_fh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_fuh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_fvh, (nx+2)*(ny+2) * sizeof ( float )));

  CHECK(hipMalloc((void **)&d_gh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_guh, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_gvh, (nx+2)*(ny+2) * sizeof ( float )));

  CHECK(hipMalloc((void **)&d_hm, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_uhm, (nx+2)*(ny+2) * sizeof ( float )));
  CHECK(hipMalloc((void **)&d_vhm, (nx+2)*(ny+2) * sizeof ( float )));

  // ************************************************ INITIAL CONDITIONS ************************************************ //

  printf ( "\n" );
  printf ( "SHALLOW_WATER_2D\n" );
  printf ( "\n" );

  // set initial time & step counter
  // set time to zero and step counter to zero
  programRuntime = 0.0f;
  k = 0;

  // Apply the initial conditions.
  initialConditions(nx, ny, dx, dy, x_length, x, y, h, uh, vh);

  // Write initial condition to a file
  writeResults(h, uh, vh, x, y, programRuntime, nx, ny);

  // Move data to the device for calculations
  CHECK(hipMemcpy(d_h, h, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_uh, uh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_vh, vh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyHostToDevice));

  // ******************************************************************** COMPUTATION SECTION ******************************************************************** //

  // start timer
  auto start_time = std::chrono::steady_clock::now();

  // Initialize timing variables
  auto last_trigger = std::chrono::steady_clock::now();
  std::chrono::milliseconds interval_time_ms(3); // 1ms interval

  while (programRuntime < finalRuntime) // time loop begins
  {
    // Take a time step and increase step counter
    programRuntime = programRuntime + dt;
    k++;

    // **** COMPUTE FLUXES ****
    computeFluxesGPU<<<gridSize, blockSize>>>(d_h, d_uh, d_vh, d_fh, d_fuh, d_fvh, d_gh, d_guh, d_gvh, nx, ny);
    
    // **** COMPUTE VARIABLES ****
    computeVariablesGPU<<<gridSize, blockSize>>>(d_hm, d_uhm, d_vhm, d_fh, d_fuh, d_fvh, d_gh, d_guh, d_gvh, d_h, d_uh, d_vh, lambda_x, lambda_y, nx, ny);

    // **** UPDATE VARIABLES ****
    updateVariablesGPU<<<gridSize, blockSize>>>(d_h, d_uh, d_vh, d_hm, d_uhm, d_vhm, nx, ny);

    // **** APPLY BOUNDARY CONDITIONS ****
    applyBoundaryConditionsGPU<<<gridSize, blockSize>>>(d_h, d_uh, d_vh, nx, ny, 3);

    // Timing check using chrono
    auto now = std::chrono::steady_clock::now();
    
    if (now - last_trigger >= interval_time_ms)
    {
      // Update timing checkpoint
      last_trigger = now;

      // Copy height, x-momentum, and y-momentum from device to host
      CHECK(hipMemcpy(h, d_h, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyDeviceToHost));
      CHECK(hipMemcpy(uh, d_uh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyDeviceToHost));
      CHECK(hipMemcpy(vh, d_vh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyDeviceToHost));

      // Randomly decide whether to generate a drop
      randNumber = rand() % 10;

      if (randNumber % 2 == 0) // Even numbers (0, 2, 4, 6, 8)
      {
      generateDrops(nx, ny, x, y, h, uh, vh);
      }

      // Copy updated water height, x-momentum, and y-momentum back to device
      CHECK(hipMemcpy(d_h, h, (nx+2)*(ny+2) * sizeof (float), hipMemcpyHostToDevice));
      CHECK(hipMemcpy(d_uh, uh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyHostToDevice));
      CHECK(hipMemcpy(d_vh, vh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyHostToDevice));
    }

  } // end time loop

  // stop timer
  auto end_time = std::chrono::steady_clock::now();
  std::chrono::duration<double> time_elapsed = end_time - start_time;

  // Print out the results
  printf("Problem size: %d, time steps taken: %d,  elapsed time: %f s\n", nx, k, time_elapsed);

  // ******************************************************************** POSTPROCESSING ******************************************************************** //

  // Move data back to the host
  CHECK(hipMemcpy(h, d_h, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyDeviceToHost));
  CHECK(hipMemcpy(uh, d_uh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyDeviceToHost));
  CHECK(hipMemcpy(vh, d_vh, (nx+2)*(ny+2) * sizeof ( float ), hipMemcpyDeviceToHost));

  writeResults(h, uh, vh, x, y, programRuntime, nx, ny);

  // ******************************************************************** DEALLOCATE MEMORY ******************************************************************** //

  //Free device memory.
  CHECK(hipFree(d_h));
  CHECK(hipFree(d_uh));
  CHECK(hipFree(d_vh));

  CHECK(hipFree(d_fh));
  CHECK(hipFree(d_fuh));
  CHECK(hipFree(d_fvh));

  CHECK(hipFree(d_gh));
  CHECK(hipFree(d_guh));
  CHECK(hipFree(d_gvh));

  CHECK(hipFree(d_hm));
  CHECK(hipFree(d_uhm));
  CHECK(hipFree(d_vhm));

  CHECK(hipFree(d_x));
  CHECK(hipFree(d_y));

  // Free host memory.
  free ( h );
  free ( uh );
  free ( vh ); 

  free ( fh );
  free ( fuh );
  free ( fvh );

  free ( gh );
  free ( guh );
  free ( gvh ); 

  free ( x );
  free ( y );

  // Terminate.
  printf ( "\n" );
  printf ( "SHALLOW_WATER_2D:\n" );
  printf ( "  Normal end of execution.\n" );
  printf ( "\n" );

  return 0;
}
// ******************************************************************************************************************************************** //