#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 4
#define HALO 1
#define BLOCK_WITH_HALO (BLOCK_SIZE + 2 * HALO)
#define GRID_DIM 3

#define IDX2D(i, j, stride) ((i) * (stride) + (j))

__device__ int SH_ID(int i, int j, int stride) {
    return i * stride + j;
}
__device__ int ID_2D(int i, int j, int stride) {
    return i * stride + j;
}

__device__ void haloExchange(float* sh_h, const float* h, int i, int j, int local_i, int local_j, int nx, int ny, int blockDim_x, int blockDim_y) 
{
    int global_stride = nx + 2;
    int sh_stride = blockDim_x + 2;

    // LEFT
    if (local_j == 1) {
        int gid = ID_2D(i, j - 1, global_stride);
        int lid = SH_ID(local_i, local_j - 1, sh_stride);
        if (j - 1 >= 0) sh_h[lid] = h[gid];
    }

    // RIGHT
    if (local_j == blockDim_x) {
        int gid = ID_2D(i, j + 1, global_stride);
        int lid = SH_ID(local_i, local_j + 1, sh_stride);
        if (j + 1 < nx + 2) sh_h[lid] = h[gid];
    }

    // BOTTOM
    if (local_i == 1) {
        int gid = ID_2D(i - 1, j, global_stride);
        int lid = SH_ID(local_i - 1, local_j, sh_stride);
        if (i - 1 >= 0) sh_h[lid] = h[gid];
    }

    // TOP
    if (local_i == blockDim_y) {
        int gid = ID_2D(i + 1, j, global_stride);
        int lid = SH_ID(local_i + 1, local_j, sh_stride);
        if (i + 1 < ny + 2) sh_h[lid] = h[gid];
    }
}

__global__ void testHaloKernel(float *h, float *h_result, int nx, int ny) {
    int global_i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int global_j = blockIdx.x * blockDim.x + threadIdx.x + 1;

    int local_i = threadIdx.y + 1;
    int local_j = threadIdx.x + 1;

    extern __shared__ float sh_h[];

    int global_stride = nx + 2;
    int sh_stride = blockDim.x + 2;

    int gid = ID_2D(global_i, global_j, global_stride);
    int lid = SH_ID(local_i, local_j, sh_stride);

    // Load interior value into shared memory
    sh_h[lid] = h[gid];
    __syncthreads();

    // Perform halo exchange
    haloExchange(sh_h, h, global_i, global_j, local_i, local_j, nx, ny, blockDim.x, blockDim.y);
    __syncthreads();

    // === DEBUG BLOCK-LEVEL SHARED MEMORY PRINT ===
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        printf("Block (%d, %d):\n", blockIdx.x, blockIdx.y);
        for (int li = 0; li < blockDim.y + 2; ++li) {
            for (int lj = 0; lj < blockDim.x + 2; ++lj) {
                int lid_dbg = SH_ID(li, lj, sh_stride);
                printf("%5.1f ", sh_h[lid_dbg]);
            }
            printf("\n");
        }
        printf("\n");
    }
    __syncthreads();

    // Write interior result
    h_result[gid] = sh_h[lid];

    // Write halo edges
    if (local_j == 1) {
        int gid_left = ID_2D(global_i, global_j - 1, global_stride);
        int lid_left = SH_ID(local_i, local_j - 1, sh_stride);
        if (global_j - 1 >= 0) h_result[gid_left] = sh_h[lid_left];
    }
    if (local_j == blockDim.x) {
        int gid_right = ID_2D(global_i, global_j + 1, global_stride);
        int lid_right = SH_ID(local_i, local_j + 1, sh_stride);
        if (global_j + 1 < nx + 2) h_result[gid_right] = sh_h[lid_right];
    }
    if (local_i == 1) {
        int gid_bottom = ID_2D(global_i - 1, global_j, global_stride);
        int lid_bottom = SH_ID(local_i - 1, local_j, sh_stride);
        if (global_i - 1 >= 0) h_result[gid_bottom] = sh_h[lid_bottom];
    }
    if (local_i == blockDim.y) {
        int gid_top = ID_2D(global_i + 1, global_j, global_stride);
        int lid_top = SH_ID(local_i + 1, local_j, sh_stride);
        if (global_i + 1 < ny + 2) h_result[gid_top] = sh_h[lid_top];
    }
}

int main() {
    const int nx = BLOCK_SIZE * GRID_DIM;
    const int ny = BLOCK_SIZE * GRID_DIM;
    const int total_size = (nx + 2) * (ny + 2);

    float *h_h = (float*)malloc(total_size * sizeof(float));
    float *h_result = (float*)malloc(total_size * sizeof(float));

    for (int i = 0; i < ny + 2; ++i) {
        for (int j = 0; j < nx + 2; ++j) {
            int block_id = (i - 1) / BLOCK_SIZE * GRID_DIM + (j - 1) / BLOCK_SIZE;
            h_h[IDX2D(i, j, nx + 2)] = (i == 0 || j == 0 || i == ny + 1 || j == nx + 1) ? -1.0f : block_id;
        }
    }

    float *d_h, *d_result;
    hipMalloc(&d_h, total_size * sizeof(float));
    hipMalloc(&d_result, total_size * sizeof(float));

    hipMemcpy(d_h, h_h, total_size * sizeof(float), hipMemcpyHostToDevice);

    // Print before halo exchange
    hipMemcpy(h_result, d_h, total_size * sizeof(float), hipMemcpyDeviceToHost);
    printf("Before halo exchange:\n");
    for (int i = 0; i < ny + 2; ++i) {
        for (int j = 0; j < nx + 2; ++j) {
            printf("%5.1f ", h_result[IDX2D(i, j, nx + 2)]);
        }
        printf("\n");
    }

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim(GRID_DIM, GRID_DIM);
    size_t shmem_size = (BLOCK_SIZE + 2) * (BLOCK_SIZE + 2) * sizeof(float);

    testHaloKernel<<<gridDim, blockDim, shmem_size>>>(d_h, d_result, nx, ny);
    hipMemcpy(h_result, d_result, total_size * sizeof(float), hipMemcpyDeviceToHost);

    printf("After halo exchange:\n");
    for (int i = 0; i < ny + 2; ++i) {
        for (int j = 0; j < nx + 2; ++j) {
            printf("%5.1f ", h_result[IDX2D(i, j, nx + 2)]);
        }
        printf("\n");
    }

    hipFree(d_h);
    hipFree(d_result);
    free(h_h);
    free(h_result);
    return 0;
}
